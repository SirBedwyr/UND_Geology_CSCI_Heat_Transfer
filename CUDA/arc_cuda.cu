#include "hip/hip_runtime.h"
/**
 * Performs a finite difference heat flow 
 * simulation using conduction and convection.
 * Uses CUDA to perform calculations on a GPGPU
 */
#define _USE_MATH_DEFINES
#ifdef _WIN32
#define NOMINMAX //FYI need to disable min/max macro in windows.h
#include <windows.h>
#endif

#ifdef DISPLAY
#ifdef __APPLE__
#  include <OpenGL/gl.h>
#  include <OpenGL/glu.h>
#  include <GLUT/glut.h>
#else
#  include <GL/GL.h>
#  include <GL/GLU.h>
#  include <GL/glut.h>
#endif
#endif

#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <iomanip>
#include <limits>
#include <cmath>

/**
 * Time_step (seconds/yr) divided by the product of density and heat capacity.
 * The value for the density is 2200 kg/m^3 and heat capacity is 1000 kJ/kg K.
 */
#define QFAC 14.33              //Description is defined in the previous comment
#define DTC 0.25                //
#define OUT_PRECISION 10        //Number of digits to print after the decimal place for floating point values
#define INDEX_WIDTH 2           //The number of characters to print and read for each conduction and convection code
#define REAL double             //The precision of the model.
#define FREEMEM 100000000       //Amount of memory to leave free on the GPU in bytes


using std::cerr;
using std::cin;
using std::cout;
using std::endl;
using std::string;
using std::ofstream;
using std::ifstream;
using std::ostringstream;
using std::setw;
using std::right;
using std::left;
using std::fixed;
using std::scientific;
using std::setprecision;
using std::setfill;
using std::ios;
using std::numeric_limits;
using std::streamsize;
using std::max;
using std::flush;

void save_surfer();
void save_model_state();
void conduction();
void convection();
void PressEnterToContinue();
REAL find_max_temp_diff();
void update_moving_sources();
void find_loc_index(REAL x_loc, REAL y_loc, REAL z_loc, int *index);


//Cuda specific variables
dim3 dimBlock;		              //Block dimensions for the kernel call
dim3 dimGrid;		              //Grid dimensions for the kernel call
hipError_t error;			      //CUDA error variable
hipDeviceProp_t deviceProp;        //CUDA device properties

//Conduction code specific variables
int *cond_codes;                  //The unmodified conduction codes as read from the input file
int *cond_hp_index;               //The conduction index for the radioactive heat production array
int *cond_tc_index;               //The conduction index for the thermal conductivity array
int *use_cond;                    //Flag to indicate if conduction occurs for a given cell
REAL DHF;                         //

//Device specific conduction variables
int *dev_cond_codes;              //The unmodified conduction codes as read from the input file
int *dev_cond_hp_index;           //The conduction index for the radioactive heat production array
int *dev_cond_tc_index;           //The conduction index for the thermal conductivity array
int *dev_use_cond;


 //Convection code specific variables
int *conv_codes;                  //The convection codes as read from the input file
int *conv_min_temp_index;         //The convection index for the minimum temp for convection array
int *conv_direction;              //The direction of convection following the direction matrix in the previous comment
int *conv_vel_index;              //The convection index for the velocity array
int *conv_fluid_index;            //The convection index for the fluid heat capacity array
int *conv_rock_index;             //The convection index for the rock heat capacity array
int num_conv_loops;               //The number of convection updates to perform per time step
REAL time_inc;                    //The amount of time increment per convection loop

//Device specific convection variables
int *dev_conv_codes;              //The convection codes as read from the input file
int *dev_conv_min_temp_index;     //The convection index for the minimum temp for convection array
int *dev_conv_direction;          //The direction of convection following the direction matrix in the previous comment
int *dev_conv_vel_index;          //The convection index for the velocity array
int *dev_conv_fluid_index;        //The convection index for the fluid heat capacity array
int *dev_conv_rock_index;         //The convection index for the rock heat capacity array

//File names
string source_filename;           //The input files name with extension
string output_filename;           //The output state files name with extension
string output_su_filename;        //The output surfer files name with extension

//Input file variables
string title;                     //The title of the input file
int using_convection = -1;        //Indicates if convection is being used
REAL *temp;                       //The current temperature array
int num_rows;                     //The number of rows for the simulation
int num_cols;                     //The number of columns for the simulation
int num_slices;                   //Total number of slices to form the 3d simulation (one 'slice' has dimension rows x columns)
REAL *dim_x;                      //The dimensions of each column in the x direction
REAL *dim_y;                      //The dimensions of each row in the y direction
REAL *dim_z;                      //The dimensions of each row in the z direction
REAL *dist_x;                     //The distance from the origin to the center of a column for a given column index in the x direction
REAL *dist_y;                     //The distance from the origin to the center of a row for a given row index in the y direction
REAL *dist_z;                     //The distance from the origin to the center of a slice for a given slice index in the z direction
REAL max_dist_x;                  //The maximum x distance
REAL max_dist_y;                  //The maximum y distance
REAL max_dist_z;                  //The maximum z distance
REAL chf;                         //Constant Heat flow at base of model in mW M^2
REAL initial_time;                //The initial starting time of the model
int num_hp;                       //The number of heat production values
int num_tcd;                      //The number of thermal conductivity difference values
int num_hcf;                      //The number of fluid heat capacity values
int num_hcr;                      //The number of rock heat capacity values
int num_mtc;                      //The number of minimum convection temperature values
int num_vel;                      //The number of convection velocities
REAL *heat_production_values;     //The radioactive heat production values array used in conduction calculations
REAL *thermal_conduct_diff;       //The thermal conductivity difference array used in conduction calculations
REAL *heat_capac_fluid;           //The fluid heat capacity array used in convection calculations
REAL *heat_capac_rock;            //The rock heat capacity array used in convection calculations
REAL *min_temp_conv;              //The minimum temperature required for convection
REAL *vel;                        //The velocity array used for convection calculations

//Device specific variables
REAL *dev_temp;                   //The current temperature array
REAL *dev_dim_x;                  //The dimensions of each column in the x direction
REAL *dev_dim_y;                  //The dimensions of each row in the y direction
REAL *dev_dim_z;                  //The dimensions of each row in the y direction
REAL *dev_dist_x;                 //The dimensions of each row in the y direction
REAL *dev_dist_y;                 //The dimensions of each row in the y direction
REAL *dev_dist_z;                 //The dimensions of each row in the y direction
REAL *dev_heat_production_values; //The radioactive heat production values array used in conduction calculations
REAL *dev_thermal_conduct_diff;   //The thermal conductivity difference array used in conduction calculations
REAL *dev_heat_capac_fluid;       //The fluid heat capacity array used in convection calculations
REAL *dev_heat_capac_rock;        //The rock heat capacity array used in convection calculations
REAL *dev_min_temp_conv;          //The minimum temperature required for convection
REAL *dev_vel;                    //The velocity array used for convection calculations


//Moving source variables
int using_moving_source = -1;     //Indicates if a moving source is being used
int num_mvsrc = -1;               //The number of moving sources
REAL *mvsrc_x;                    //The x location of the moving sources
REAL *mvsrc_y;                    //The y location of the moving sources
REAL *mvsrc_z;                    //The z location of the moving sources
REAL *mvsrc_offset_x;             //The size of the moving sources in the x direction
REAL *mvsrc_offset_y;             //The size of the moving sources in the y direction
REAL *mvsrc_offset_z;             //The size of the moving sources in the z direction
REAL *mvsrc_vel_x;                //The x component of the moving sources velocity vectors
REAL *mvsrc_vel_y;                //The y component of the moving sources velocity vectors
REAL *mvsrc_vel_z;                //The z component of the moving sources velocity vectors
REAL *mvsrc_accel_x;              //The x component of the moving sources acceleration vectors
REAL *mvsrc_accel_y;              //The y component of the moving sources acceleration vectors
REAL *mvsrc_accel_z;              //The z component of the moving sources acceleration vectors
REAL *mvsrc_temp;                 //The temperature of the moving sources
int *mvsrc_valid;                 //Indicates if a moving source is valid

//Time specific variables
REAL sim_time;                    //The current time of the simulation
REAL tic;                         //Time variable used in convection calculations
REAL time_step = -1;              //The amount of time that passes between each update of the simulation, time step
REAL run_time = -1;               //The run time of the simulation

//Global variables
int save_state = -1;              //Indicates if the model should save the current state at each screen update
int save_result = -1;             //Indicates if the model should save the final result of the simulaiton
int use_tolerance = -1;           //Indicates if the model should stop once a user specified tolerance is met for temperature change
REAL max_vel;                     //The maximum convection velocity of the velocity array
REAL min_row_dim;                 //The minimum y dimension of each cell of the simulation
REAL min_col_dim;                 //The minimum x dimension of each cell of the simulation
REAL min_slice_dim;               // The minimum z dimension of each cell in the simulation
REAL thermal_time_constant;       //The thermal time constant of the model, used in the selection of the run time
REAL *next_temp;                  //The next temperature array
REAL max_thermal_conduct_diff;    //The maximum thermal conductivity difference
REAL min_thermal_conduct_diff;    //The minimum thermal conductivity difference
int num_loops = -1;               //The number of loops between screen updates
int su_num_width;                 //The number of characters for the slice number in the output surfer filenames
unsigned long long count = 0;	  //The current loop
REAL tolerance = -1;              //The maximum difference required for the model to stop
REAL max_temp_diff;               //The maximum temperature difference between the current and next temperature arrays
unsigned long long num_cells;     //The number of cells in the simulation
REAL *dev_next_temp;              //The next temperature array


/**
 * Deallocates all allocated memory used by the program
 */
void deallocate_memory() {
    //Deletes allocated memory
    delete[] dim_x;
    delete[] dim_y;
    delete[] dim_z;
    delete[] dist_x;
    delete[] dist_y;
    delete[] dist_z;
    delete[] temp;
    delete[] next_temp;
    delete[] cond_codes;
    delete[] cond_hp_index;
    delete[] cond_tc_index;
    delete[] use_cond;
    delete[] heat_production_values;
    delete[] thermal_conduct_diff;
    if(using_convection == 1) {
        delete[] conv_codes;
        delete[] conv_min_temp_index;
        delete[] conv_direction;
        delete[] conv_vel_index;
        delete[] conv_fluid_index;
        delete[] conv_rock_index;
        delete[] heat_capac_fluid;
        delete[] heat_capac_rock;
        delete[] min_temp_conv;
        delete[] vel;
    }
    if(using_moving_source == 1) {
        delete[] mvsrc_x;
        delete[] mvsrc_y;
        delete[] mvsrc_z;
        delete[] mvsrc_offset_x;
        delete[] mvsrc_offset_y;
        delete[] mvsrc_offset_z;
        delete[] mvsrc_vel_x;
        delete[] mvsrc_vel_y;
        delete[] mvsrc_vel_z;
        delete[] mvsrc_accel_x;
        delete[] mvsrc_accel_y;
        delete[] mvsrc_accel_z;
        delete[] mvsrc_temp;
        delete[] mvsrc_valid;
    }
}

void deallocate_cuda_memory() {
    hipFree(dev_temp);
    hipFree(dev_next_temp);
    hipFree(dev_dim_x);
    hipFree(dev_dim_y);
    hipFree(dev_dim_z);
    hipFree(dev_dist_x);
    hipFree(dev_dist_y);
    hipFree(dev_dist_z);
    hipFree(dev_cond_codes);
    hipFree(dev_cond_hp_index);
    hipFree(dev_cond_tc_index);
    hipFree(dev_use_cond);
    hipFree(dev_heat_production_values);
    hipFree(dev_thermal_conduct_diff);
    if(using_convection) {
        hipFree(dev_heat_capac_fluid);
        hipFree(dev_heat_capac_rock);
        hipFree(dev_min_temp_conv);
        hipFree(dev_vel);
        hipFree(dev_conv_codes);
        hipFree(dev_conv_min_temp_index);
        hipFree(dev_conv_direction);
        hipFree(dev_conv_vel_index);
        hipFree(dev_conv_fluid_index);
        hipFree(dev_conv_rock_index);
    }
}

#ifdef DISPLAY
//Display variables
int display_mode = -1;
int array_size;
REAL min_temp;
REAL max_temp;
REAL layer_min_temp;
REAL layer_max_temp;
float *color_field;
float transparency = 1.0f;
int current_slice = 0;

/**
 *  Parameters to control the camera angle so we can move where we're looking at
 *  the simulation from with the mouse. Kept for debugging.
 */
/*
int     ox                  = 0;
int     oy                  = 0;
int     buttonState         = 0; 
float   camera_trans[]      = {0, -0.2, -10};
float   camera_rot[]        = {0, 0, 0};
float   camera_trans_lag[]  = {0, -0.2, -10};
float   camera_rot_lag[]    = {0, 0, 0};
const float inertia         = 0.1f;
*/

/*
 * Sets max and min temperature values for simulation
 */
void array_minmax() {
	min_temp=temp[0][0][0];
	max_temp=temp[0][0][0];
    for(int k=0; k<num_slices; k++) {
		for (int i=0; i<num_rows; i++) {
			for(int j=0; j<num_cols; j++) {
				if(temp[i][j][k]<min_temp)
					min_temp = temp[i][j][k];
				if(temp[i][j][k]>max_temp)
					max_temp = temp[i][j][k];
			}
		}
	}
}

/**
 * Updates max_temp with the maximum temp of
 * the current slice.
 */
void array_max() {
    max_temp=temp[0][0][0];
    for(int i = 0; i < num_rows; i++) {
        for(int j = 0; j < num_cols; j++) {
            if(temp[i][j][current_slice] > max_temp) {
                max_temp = temp[i][j][current_slice];
            }
        }
    }
}

/* 
 * 3D to 1D indexing
 */
static int POSITION(int x, int y) {
	return (x*num_cols)+y;
}


/*
 * Colormap algorithm reproduces Matlab's RGB "Jet" plate
 * Concept based on: http://paulbourke.net/texture_colour/colourspace/ (11/21/12)
 */
void jet_color_set(int x, int y, int z) {
	REAL current_temp = temp[x][y][z];
	REAL delta_temp = max_temp - min_temp;
	
	if(current_temp < min_temp)
		current_temp = min_temp;
	if(current_temp > max_temp)
		current_temp = max_temp;

	if(current_temp < (min_temp + 0.25 * delta_temp)) {
		color_field[POSITION(x,y) * 3] = (GLfloat)0.0;	
		color_field[POSITION(x,y) * 3 + 1] = (GLfloat)(4*(current_temp - min_temp)/delta_temp);
		color_field[POSITION(x,y) * 3 + 2] = (GLfloat)1.0;
	}
	else if(current_temp < (min_temp + 0.5 * delta_temp)) {
		color_field[POSITION(x,y) * 3] = (GLfloat)0.0;	
		color_field[POSITION(x,y) * 3 + 1] = (GLfloat)1.0;	
		color_field[POSITION(x,y) * 3 + 2] = (GLfloat)(1.0 + 4 * (min_temp + 0.25 * delta_temp - current_temp) / delta_temp);
	}
	else if(current_temp < (min_temp + 0.75 * delta_temp)) {
		color_field[POSITION(x,y) * 3] = (GLfloat)(4 * (current_temp - min_temp - 0.5 * delta_temp) / delta_temp);	
		color_field[POSITION(x,y) * 3 + 1] = (GLfloat)1.0;
		color_field[POSITION(x,y) * 3 + 2] = (GLfloat)0.0;
	}
	else {
		color_field[POSITION(x,y) * 3] = (GLfloat)1.0;	
		color_field[POSITION(x,y) * 3 + 1] = (GLfloat)(1.0 + 4 * (min_temp + 0.75 * delta_temp - current_temp) / delta_temp);
		color_field[POSITION(x,y) * 3 + 2] = (GLfloat)0.0;
	}
}


/*
 * Draw temp surface via 1x1 faces. Dimensions constant for simplicity.
 * Originally drew cubes from Robert Bergmans voxel display code.
 */
void draw_cube(int x, int y, int z) {
	if(z == current_slice) {
		transparency = 1.0f;
	}
	else {
		transparency = 0.3f;
	}
	glBegin(GL_TRIANGLES);
		//front
		glColor4f(	color_field[POSITION(x,y) * 3],
					color_field[POSITION(x,y) * 3 + 1],
					color_field[POSITION(x,y) * 3 + 2],
					transparency);
		glVertex3f(0.0f,0.0f,1.0f);//5
		glVertex3f(1.0f,0.0f,1.0f);//6
		glVertex3f(0.0f,-1.0f,1.0f);//8
		
		glColor4f(	color_field[POSITION(x,y) * 3],
					color_field[POSITION(x,y) * 3 + 1],
					color_field[POSITION(x,y) * 3 + 2],
					transparency);
		glVertex3f(0.0f,-1.0f,1.0f);//8
		glVertex3f(1.0f,0.0f,1.0f);//6
		glVertex3f(1.0f,-1.0f,1.0f);//7

		//top
		glColor4f(	color_field[POSITION(x,y) * 3],
					color_field[POSITION(x,y) * 3 + 1],
					color_field[POSITION(x,y) * 3 + 2],
					transparency);
		glVertex3f(0.0f,0.0f,0.0f);//1
		glVertex3f(1.0f,0.0f,0.0f);//2
		glVertex3f(0.0f,0.0f,1.0f);//5
		
		glColor4f(	color_field[POSITION(x,y) * 3],
					color_field[POSITION(x,y) * 3 + 1],
					color_field[POSITION(x,y) * 3 + 2],
					transparency);
		glVertex3f(0.0f,0.0f,1.0f);//5
		glVertex3f(1.0f,0.0f,0.0f);//2
		glVertex3f(1.0f,0.0f,1.0f);//6
	
	
	//QUADS code left in case we need it later	
	/*
	glBegin(GL_QUADS);
		//front
		glColor4f(	color_field[POSITION(x,y) * 3],
					color_field[POSITION(x,y) * 3 + 1],
					color_field[POSITION(x,y) * 3 + 2],
					transparency);
		glVertex3f(0.0f,0.0f,1.0f);//5
		glVertex3f(1.0f,0.0f,1.0f);//6
		glVertex3f(1.0f,-1.0f,1.0f);//7
		glVertex3f(0.0f,-1.0f,1.0f);//8

		//top
		glColor4f(	color_field[POSITION(x,y) * 3],
					color_field[POSITION(x,y) * 3 + 1],
					color_field[POSITION(x,y) * 3 + 2],
					transparency);
		glVertex3f(0.0f,0.0f,0.0f);//1
		glVertex3f(1.0f,0.0f,0.0f);//2
		glVertex3f(1.0f,0.0f,1.0f);//6
		glVertex3f(0.0f,0.0f,1.0f);//5

		*//*//left
		glColor4f(	color_field[POSITION(x,y) * 3],
					color_field[POSITION(x,y) * 3 + 1],
					color_field[POSITION(x,y) * 3 + 2],
					transparency);
		glVertex3f(0.0f,0.0f,0.0f);//1
		glVertex3f(0.0f,0.0f,1.0f);//5
		glVertex3f(0.0f,-1.0f,1.0f);//8
		glVertex3f(0.0f,-1.0f,0.0f);//4

		//right
		glColor4f(	color_field[POSITION(x,y) * 3],
					color_field[POSITION(x,y) * 3 + 1],
					color_field[POSITION(x,y) * 3 + 2],
					transparency);
		glVertex3f(1.0f,0.0f,0.0f);//2
		glVertex3f(1.0f,0.0f,1.0f);//6
		glVertex3f(1.0f,-1.0f,1.0f);//7
		glVertex3f(1.0f,-1.0f,0.0f);//3

		//bottom
		glColor4f(	color_field[POSITION(x,y) * 3],
					color_field[POSITION(x,y) * 3 + 1],
					color_field[POSITION(x,y) * 3 + 2],
					transparency);
		glVertex3f(0.0f,-1.0f,0.0f);//4
		glVertex3f(1.0f,-1.0f,0.0f);//3
		glVertex3f(1.0f,-1.0f,1.0f);//7
		glVertex3f(0.0f,-1.0f,1.0f);//8

		//back
		glColor4f(	color_field[POSITION(x,y) * 3],
					color_field[POSITION(x,y) * 3 + 1],
					color_field[POSITION(x,y) * 3 + 2],
					transparency);
		glVertex3f(0.0f,0.0f,0.0f);//1
		glVertex3f(1.0f,0.0f,0.0f);//2
		glVertex3f(1.0f,-1.0f,0.0f);//3
		glVertex3f(0.0f,-1.0f,0.0f);//4
		*/

	glEnd();
}


/*
 *Draw all HUD/Overlay graphics. Quads for temp scale are hardcoded to Jet color map. Any new color map will require changes.
 */
void displayOverlay(){	
	int windowWidth = glutGet(GLUT_WINDOW_WIDTH);
	int windowHeight = glutGet(GLUT_WINDOW_HEIGHT);
    
	glMatrixMode( GL_PROJECTION );
    glPushMatrix();
        glLoadIdentity();
        glOrtho(0.0f,windowWidth,windowHeight,0.0f,0.0f,1.0f);

        glMatrixMode( GL_MODELVIEW );
        glPushMatrix();
            glLoadIdentity();
			glBegin( GL_QUADS );

				glColor3f( 0.0f, 0.0f, 1.0f );
                glVertex2f( (GLfloat)(windowWidth/2-75), 20.0f );
                glVertex2f( (GLfloat)(windowWidth/2-45), 20.0f );
                glVertex2f( (GLfloat)(windowWidth/2-45), 50.0f );
                glVertex2f( (GLfloat)(windowWidth/2-75), 50.0f );

				glColor3f( 0.0f, 1.0f, 1.0f );
                glVertex2f( (GLfloat)(windowWidth/2-45), 20.0f );
                glVertex2f( (GLfloat)(windowWidth/2-15), 20.0f );
                glVertex2f( (GLfloat)(windowWidth/2-15), 50.0f );
                glVertex2f( (GLfloat)(windowWidth/2-45), 50.0f );

				glColor3f( 0.0f, 1.0f, 0.0f );
                glVertex2f( (GLfloat)(windowWidth/2-15), 20.0f );
                glVertex2f( (GLfloat)(windowWidth/2+15), 20.0f );
                glVertex2f( (GLfloat)(windowWidth/2+15), 50.0f );
                glVertex2f( (GLfloat)(windowWidth/2-15), 50.0f );

				glColor3f( 1.0f, 1.0f, 0.0f );
                glVertex2f( (GLfloat)(windowWidth/2+15), 20.0f );
                glVertex2f( (GLfloat)(windowWidth/2+45), 20.0f );
                glVertex2f( (GLfloat)(windowWidth/2+45), 50.0f );
                glVertex2f( (GLfloat)(windowWidth/2+15), 50.0f );

				glColor3f( 1.0f, 0.0f, 0.0f );
                glVertex2f( (GLfloat)(windowWidth/2+45), 20.0f );
                glVertex2f( (GLfloat)(windowWidth/2+75), 20.0f );
                glVertex2f( (GLfloat)(windowWidth/2+75), 50.0f );
                glVertex2f( (GLfloat)(windowWidth/2+45), 50.0f );

            glEnd();
        glPopMatrix();
		glPushMatrix();
			glLoadIdentity();
			ostringstream str1;

			str1 << "Min Temp <                                          > Max Temp";
			glColor3f(1.0f, 1.0f, 1.0f); 
			glRasterPos2f((GLfloat)(windowWidth/2-150),35.0f);
			glutBitmapString(GLUT_BITMAP_HELVETICA_12, (const unsigned char *)str1.str().c_str());
			str1.str("");
			str1.clear();
			
			str1 << setw(4) << min_temp;
			glRasterPos2f((GLfloat)(windowWidth/2-100), 65.0f);
			glutBitmapString(GLUT_BITMAP_HELVETICA_12, (const unsigned char *)str1.str().c_str());
			str1.str("");
			str1.clear();

			str1 << setw(4) << (max_temp+min_temp)/2;
			glRasterPos2f((GLfloat)(windowWidth/2-20), 65.0f);
			glutBitmapString(GLUT_BITMAP_HELVETICA_12, (const unsigned char *)str1.str().c_str());
			str1.str("");
			str1.clear();


			str1 << setw(4) << max_temp;
			glRasterPos2f((GLfloat)(windowWidth/2+60), 65.0f);
			glutBitmapString(GLUT_BITMAP_HELVETICA_12, (const unsigned char *)str1.str().c_str());
			str1.str("");
			str1.clear();


		glPopMatrix();
        glPushMatrix();
            glLoadIdentity();
            ostringstream str;

			str << "Time Interval:" << endl;
			if(using_convection) {
				str << "Conv. Time Interval:" << endl;
				str << "Num Conv. Loops:" << endl;
			}

			str << endl << "Loop Total:" << endl;
			str << "Sim Time:" << endl;
			str << "Cum. Sim Time:" << endl << endl;
		
			str << "Model Dimensions:" << endl;
			str << "Current Slice:" << endl;
			str << "CHF:" << endl;
			
			glColor3f(1.0f, 1.0f, 1.0f); 
			glRasterPos2f(10.0f,(GLfloat)(windowHeight*3.0/4.0));
			glutBitmapString(GLUT_BITMAP_HELVETICA_12, (const unsigned char *)str.str().c_str());
		

			str.str("");
			str.clear();
		
			str << setprecision(4) << scientific;
			str << time_step << endl;
			if(using_convection) {
				str << time_inc << endl;
				str << num_conv_loops << endl;
			}

			str << endl << count << endl;
			str << sim_time << endl;
			str << initial_time + sim_time << endl << endl;
			
			str << num_rows << " X " << num_cols << " X " << num_slices << endl;
			str << current_slice+1 << endl;
			str << chf << endl;
			
			glColor3f(1.0f, 1.0f, 1.0f); 
			glRasterPos2f(150.0f,(GLfloat)(windowHeight*3.0/4.0));
			glutBitmapString(GLUT_BITMAP_HELVETICA_12, (const unsigned char *)str.str().c_str());
		            
        glPopMatrix();

    glMatrixMode( GL_PROJECTION );
    glPopMatrix();
}

/*
 * Helper function called from display3d. Broken out for readability.
 */
void display_helper() {
    array_max();
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	glEnable(GL_DEPTH_TEST);

	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();

	//  Handle the camera angle. Maintained for debugging
	/*
    for (int c = 0; c < 3; ++c)
    {
        camera_trans_lag[c] += (camera_trans[c] - camera_trans_lag[c]) * inertia;
        camera_rot_lag[c] += (camera_rot[c] - camera_rot_lag[c]) * inertia;
    }

    glTranslatef(camera_trans_lag[0], camera_trans_lag[1], camera_trans_lag[2]);
    glRotatef(camera_rot_lag[0], 1.0, 0.0, 0.0);
    glRotatef(camera_rot_lag[1], 0.0, 1.0, 0.0);
	*/

	//Draw the boundary lines
	glBegin(GL_LINES);
		glColor3f(1.0, 1.0, 1.0);

		glVertex3f(0.0f,0.0f,0.0f);
		glVertex3f((GLfloat)num_cols,0.0f,0.0f);
		glVertex3f((GLfloat)num_cols,0.0f,0.0f);
		glVertex3f((GLfloat)num_cols,(GLfloat)-num_rows,0.0f);
		glVertex3f((GLfloat)num_cols,(GLfloat)-num_rows,0.0f);
		glVertex3f(0.0f,(GLfloat)-num_rows,0.0f);
		glVertex3f(0.0f,(GLfloat)-num_rows,0.0f);
		glVertex3f(0.0f,0.0f,0.0f);

		glVertex3f(0.0f,0.0f,(GLfloat)num_slices);
		glVertex3f((GLfloat)num_cols,0.0f,(GLfloat)num_slices);
		glVertex3f((GLfloat)num_cols,0.0f,(GLfloat)num_slices);
		glVertex3f((GLfloat)num_cols,(GLfloat)-num_rows,(GLfloat)num_slices);
		glVertex3f((GLfloat)num_cols,(GLfloat)-num_rows,(GLfloat)num_slices);
		glVertex3f(0.0f,(GLfloat)-num_rows,(GLfloat)num_slices);
		glVertex3f(0.0f,(GLfloat)-num_rows,(GLfloat)num_slices);
		glVertex3f(0.0f,0.0f,(GLfloat)num_slices);

		glVertex3f(0.0f,0.0f,0.0f);
		glVertex3f(0.0f,0.0f,(GLfloat)num_slices);
		glVertex3f((GLfloat)num_cols,0.0f,0.0f);
		glVertex3f((GLfloat)num_cols,0.0f,(GLfloat)num_slices);
		glVertex3f((GLfloat)num_cols,(GLfloat)-num_rows,0.0f);
		glVertex3f((GLfloat)num_cols,(GLfloat)-num_rows,(GLfloat)num_slices);
		glVertex3f(0.0f,(GLfloat)-num_rows,0.0f);
		glVertex3f(0.0f,(GLfloat)-num_rows,(GLfloat)num_slices);
	glEnd();
	
	for(int i=0; i<num_rows; i++) {
		for (int j=0; j<num_cols; j++) {
			glPushMatrix();
			glTranslatef((GLfloat)j,(GLfloat)-i,(GLfloat)current_slice);
			jet_color_set(i,j,current_slice);
			draw_cube(i,j,current_slice);
			glPopMatrix();
		}
	}

	displayOverlay();
	glutSwapBuffers();
	glutPostRedisplay();
}

/*
 * Main simulation call during display. Makes required simulation calls e.g.- for convection, etc. 
 * Makes call to display helper function
 */
void display3D() {
	//Displays status information for the current loop
	if(count%num_loops == 0) {
        if(use_tolerance == 0) {
		    cout << setw(15) << count << setw(20) << fixed << setprecision(5) << sim_time << setw(20) << initial_time + sim_time << endl;
        }
        else {
            cout << setw(15) << count << setw(20) << fixed << setprecision(5) << sim_time << setw(20) << initial_time + sim_time << setw(20) << max_temp_diff << endl;
        }

		//Saves the current state of the simulation if the save_state flag is set
		if(save_state) {
			save_model_state();
		}
		display_helper();
	}
	if(sim_time <= run_time) {
		//Performs convection updates if the current simulation is using convection
		if(using_convection) {
			convection();
		}

		//Performs conduction calculations
		conduction();

		//Increments the simulation time and loop count
		sim_time += time_step;
		count++;
        
        if(use_tolerance == 1) {
            max_temp_diff = find_max_temp_diff();
            if(max_temp_diff < tolerance) {
                cout << "Maximum temperature change below the tolerance, stoping the simulation" << endl;
                //Saves the final result of the simulation
                if(save_state == 1 || save_result == 1) {
                    save_model_state();
                }
                save_surfer();
                cout << endl << "Simulation Complete" << endl;

                delete[] color_field;
                deallocate_memory();
                glutLeaveMainLoop();
            }
        }

        //Updates the moving source
        if(using_moving_source == 1) {
		    update_moving_sources();
        }
	}
	else {
		//Saves the final result of the simulation
		if(save_state == 1 || save_result == 1) {
			save_model_state();
		}
		save_surfer();
		cout << endl << "Simulation Complete" << endl;

		delete[] color_field;
        deallocate_memory();
		glutLeaveMainLoop();
	}
	glutPostRedisplay();
}


/*
 * This captures information when the mouse buttons are pressed.
 * Maintained for debugging.
 */
/*
void mouse_button(int button, int state, int x, int y) {
    int mods;

    if (state == GLUT_DOWN)
        buttonState |= 1<<button;
    else if (state == GLUT_UP)
        buttonState = 0;

    mods = glutGetModifiers();
    if (mods & GLUT_ACTIVE_SHIFT) 
    {
        buttonState = 2;
    } 
    else if (mods & GLUT_ACTIVE_CTRL) 
    {
        buttonState = 3;
    }

    ox = x; oy = y;

    glutPostRedisplay();
}
*/

/*
 * This captures mouse motion information.
 * Maintained for debugging
 */
/*
void mouse_move(int x, int y) {
    float dx = (float)(x - ox);
    float dy = (float)(y - oy);

    if (buttonState == 3) 
    {
        // left+middle = zoom
        camera_trans[2] += (dy / 100.0f) * 0.5f * fabs(camera_trans[2]);
    } 
    else if (buttonState & 2) 
    {
        // middle = translate
        camera_trans[0] += dx / 10.0f;
        camera_trans[1] -= dy / 10.0f;
    }
    else if (buttonState & 1) 
    {
        // left = rotate
        camera_rot[0] += dy / 5.0f;
        camera_rot[1] += dx / 5.0f;
    }

    ox = x; oy = y;
    glutPostRedisplay();
}
*/

/*
 *Standard keyboard character control
 */
void keyboard(unsigned char key, int x, int y) {
	switch(key) {
	case '-':
		if(current_slice > 0) {
			current_slice--;
			//camera_trans[1]-=0.5;//For camera mouse control, above
			//camera_trans[2]+=1.0;
			glMatrixMode(GL_PROJECTION);
			glLoadIdentity();
			gluPerspective(60, 1.77777f, 1.0, 20000.0);
            //gluLookAt(num_cols/2.0,num_rows*0.1,num_rows+current_slice,num_cols/2.0,-num_rows/3.0,current_slice,0.0,1.0,0.0);
            if(num_rows > num_cols) {
		        gluLookAt(num_cols/2.0,num_rows*0.1,num_rows+current_slice,num_cols/2.0,-num_rows/3.0,current_slice,0.0,1.0,0.0);
            }
            else {
                gluLookAt(num_cols/2.0,num_rows*0.1,num_cols+current_slice,num_cols/2.0,-num_rows/3.0,current_slice,0.0,1.0,0.0);
            }
		}
		break;
	case '+':
		if(current_slice < num_slices-1) {
			current_slice++;
			//camera_trans[1]+=0.5;//For camera mouse control, above
			//camera_trans[2]-=1.0;
			glMatrixMode(GL_PROJECTION);
			glLoadIdentity();
			gluPerspective(60, 1.77777f, 1.0, 20000.0);
			//gluLookAt(num_cols/2.0,num_rows*0.1,num_rows+current_slice,num_cols/2.0,-num_rows/3.0,current_slice,0.0,1.0,0.0);
            if(num_rows > num_cols) {
		        gluLookAt(num_cols/2.0,num_rows*0.1,num_rows+current_slice,num_cols/2.0,-num_rows/3.0,current_slice,0.0,1.0,0.0);
            }
            else {
                gluLookAt(num_cols/2.0,num_rows*0.1,num_cols+current_slice,num_cols/2.0,-num_rows/3.0,current_slice,0.0,1.0,0.0);
            }
		}
		break;
	case 'x':
		exit(0);
	default:
		break;
	}
	display_helper();
}

/*
 * Special keyboard control for arrows
 */
void keyboardSpecial(int key, int x, int y) {
	switch(key) {
	case GLUT_KEY_UP:
		if(current_slice > 0) {			
			current_slice--;
			//camera_trans[1]-=0.5;
			//camera_trans[2]+=1.0;
			glMatrixMode(GL_PROJECTION);
			glLoadIdentity();
			gluPerspective(60, 1.77777f, 1.0, 20000.0);
			//gluLookAt(num_cols/2.0,num_rows*0.1,num_rows+current_slice,num_cols/2.0,-num_rows/3.0,current_slice,0.0,1.0,0.0);
            if(num_rows > num_cols) {
		        gluLookAt(num_cols/2.0,num_rows*0.1,num_rows+current_slice,num_cols/2.0,-num_rows/3.0,current_slice,0.0,1.0,0.0);
            }
            else {
                gluLookAt(num_cols/2.0,num_rows*0.1,num_cols+current_slice,num_cols/2.0,-num_rows/3.0,current_slice,0.0,1.0,0.0);
            }
		}
		break;

	case GLUT_KEY_DOWN:
		if(current_slice < num_slices-1) {
			current_slice++;
			//camera_trans[1]+=0.5;
			//camera_trans[2]-=1.0;
			glMatrixMode(GL_PROJECTION);
			glLoadIdentity();
			gluPerspective(60, 1.77777f, 1.0, 20000.0);
			//gluLookAt(num_cols/2.0,num_rows*0.1,num_rows+current_slice,num_cols/2.0,-num_rows/3.0,current_slice,0.0,1.0,0.0);
            if(num_rows > num_cols) {
		        gluLookAt(num_cols/2.0,num_rows*0.1,num_rows+current_slice,num_cols/2.0,-num_rows/3.0,current_slice,0.0,1.0,0.0);
            }
            else {
                gluLookAt(num_cols/2.0,num_rows*0.1,num_cols+current_slice,num_cols/2.0,-num_rows/3.0,current_slice,0.0,1.0,0.0);
            }
		}
		break;
	default:
		break;
	}
	display_helper();
}
#endif

/**
 * Clears the cin buffer
 */
void clear_cin() {
	cin.clear();
	cin.ignore(numeric_limits <streamsize> ::max(), '\n' );
}

/**
 * This function waits for the user to hit enter before continuing
 */
void PressEnterToContinue() {
    cout << "Press ENTER to continue... " << flush;
    clear_cin();
}


/**
 * Swaps the temp arrays
 */
void swap_temp_array() {
    REAL *tmp;
    
    tmp = temp;
    temp = next_temp;
    next_temp = tmp;
}

void swap_temp_array_cuda() {
    REAL *tmp;
    
    tmp = dev_temp;
    dev_temp = dev_next_temp;
    dev_next_temp = tmp;
}

/**
 * Loads the input file into program memory and allocates
 * necessary memory to store the input variables
 */
void load_file() {
    ifstream source_file;       //Input file stream
    string temp_str;
    ostringstream str_conv;
    
    //Ask for the input file names and displays an error message
    //if the file does not exist
    do {
        cout << "Input File Name: ";
        cin >> source_filename;
        source_file.open(source_filename.c_str(),ios::in);
        if(!source_file.is_open()) {
            cout << "File Not found!" << endl;
        }
    } while(!source_file.is_open());
    
    //Asks the user if the state of the model should be saved every screen update
    cout << endl << "To save the state of the model every screen update enter 1, otherwise 0: ";
    while(!(cin >> save_state) || save_state < 0 || save_state > 1) {
		clear_cin();
        cout << "Incorrect input, to save the state of the model enter 1, else 0: ";
    }

    if(save_state == 0) {
        //Asks the user if the final result of the model should be saved
        cout << endl << "To save the final result of the model enter 1, otherwise 0: ";
        while(!(cin >> save_result) || save_result < 0 || save_result > 1) {
			clear_cin();
            cout << "Incorrect input, to save the final result of the model enter 1, else 0: ";
        }
    }
    
    //Ask for the state filename if the user specified that the file should be saved
    if(save_state == 1 || save_result == 1) {
        cout << "Output filename: ";
        cin >> output_filename;
    }
    
    //Asks for the DSAA surfer grid filenmae
    cout << "Surfer filename: ";
    while(!(cin >> output_su_filename) || output_su_filename.length() < 5) {
		clear_cin();
		cout << "Please enter a filename at least 5 characters in length: ";
	}

    //Loads the input file
    cout << endl << endl << "Loading Input File";
    
    //Retrieves the simulation parameters from the input file
    source_file >> num_rows >> num_cols >> num_slices >> using_convection;
    source_file >> chf >> initial_time;
    getline(source_file,title);
    getline(source_file,title);
    
    //Calculates the number of cells in the simulation
    num_cells = num_rows*num_cols*num_slices;
    
    //Calculates the dimensions of the grid and blocks
    if(num_cells <= deviceProp.maxThreadsDim[0]) {
        dimBlock.x = num_cells;
        dimGrid.x = 1;
    }
    else {
        dimBlock.x = deviceProp.maxThreadsDim[0];
        dimGrid.x = (int)(ceil(num_cells/(REAL)deviceProp.maxThreadsDim[0]));
    }
    
    //Calculates the amount of memory to be used by the program
    unsigned long long total_mem_used = num_rows*num_cols*num_slices*(2*sizeof(REAL) + 4*sizeof(int)) + sizeof(REAL)*2*(num_rows+num_cols+num_slices);
    if(using_convection) {
        total_mem_used += num_rows*num_cols*num_slices*6*sizeof(int);
    }
    
    //Exits the program if the estimated amount of memory exceeds the amount of global memory
    cout << endl << endl << "Estimated amount of memory usage: " << total_mem_used << endl;
    if(total_mem_used > deviceProp.totalGlobalMem-FREEMEM) {
        cout << "Simulation exceeds global memory limits of GPU, Exiting Program!" << endl;
        exit(1);
    }
    //Checks the total and used amount of device global memory before allocation
    size_t free_memory;		//Free memory on the device
    size_t total_memory;	//Total memory on the device
    error = hipMemGetInfo(&free_memory, &total_memory);	//Retrieves the memory information for the device
    if(error != hipSuccess) {
        cerr << endl << "Error while getting memory information" << endl;
		cerr << error << ":" << hipGetErrorString(error) << endl;
		cerr << "Exiting the Program" << endl;
		exit(0);
	}
    cout << "Free memory: "<< (unsigned int)free_memory << ", total memory: "<< (unsigned int)total_memory<<" (before initialization)" << endl;
    
    //displays parameters of the input file
    cout << endl << endl << "Total Number of Cells = " << num_cells << endl;
    cout <<  "Number of rows   = " << num_rows << endl;
    cout << "Number of cols   = " << num_cols << endl;
    cout << "Number of slices = " << num_slices << endl;
    if(using_convection == 1) {
        cout << "Using convection" << endl;
    }
    else {
        cout << "No Convection" << endl;
    }
    cout << endl << "Constant Heat Flow at Base of Model = " << chf << "mW M^2" << endl;
    chf *= 0.001;
    cout << "Model time elapsed = " << initial_time << " Years" << endl << endl;
    
    //Calculates the number of characters for the surfer file index
    str_conv << num_slices;
    su_num_width = str_conv.str().length();
    
    //Allocates memory for the conduction variables based on the previously read in simulation
    //parameters
    dim_x = new REAL[num_cols];
    dim_y = new REAL[num_rows];
    dim_z = new REAL[num_slices];
    dist_x = new REAL[num_cols];
    dist_y = new REAL[num_rows];
    dist_z = new REAL[num_slices];
    temp = new REAL[num_rows*num_cols];
    next_temp = new REAL[num_rows*num_cols];
    cond_codes = new int[num_rows*num_cols];
    cond_hp_index = new int[num_rows*num_cols];
    cond_tc_index = new int[num_rows*num_cols];
    use_cond = new int[num_rows*num_cols];
    
    //Allocates conduction specifc variables in device memory
    error = hipMalloc((void **) &dev_dim_x,num_cols*sizeof(REAL));
    error = hipMalloc((void **) &dev_dim_y,num_rows*sizeof(REAL));
    error = hipMalloc((void **) &dev_dim_z,num_slices*sizeof(REAL));
    error = hipMalloc((void **) &dev_dist_x,num_cols*sizeof(REAL));
    error = hipMalloc((void **) &dev_dist_y,num_rows*sizeof(REAL));
    error = hipMalloc((void **) &dev_dist_z,num_slices*sizeof(REAL));
    error = hipMalloc((void **) &dev_temp,num_cols*num_rows*num_slices*sizeof(REAL));
    error = hipMalloc((void **) &dev_next_temp,num_cols*num_rows*num_slices*sizeof(REAL));
    error = hipMalloc((void **) &dev_cond_codes,num_cols*num_rows*num_slices*sizeof(int));
    error = hipMalloc((void **) &dev_cond_hp_index,num_cols*num_rows*num_slices*sizeof(int));
    error = hipMalloc((void **) &dev_cond_tc_index,num_cols*num_rows*num_slices*sizeof(int));
    error = hipMalloc((void **) &dev_use_cond,num_cols*num_rows*num_slices*sizeof(int));
    if(error != hipSuccess) {
        cerr << "Unable to allocate device memory for conduction variables" << endl;
        exit(1);
    }
    
    //Reads in the starting temperatures of the simulation from the input file
    for (int k = 0; k < num_slices; k++) {
        for(int i = 0; i < num_rows; i++) {
            for(int j = 0; j < num_cols; j++) {
                source_file >> temp[i*num_cols + j];
            }
        }
        //Copies the current termperature slice to the device
        error = hipMemcpy(&dev_temp[k*num_cols*num_rows],temp,num_rows*num_cols*sizeof(REAL),hipMemcpyHostToDevice);
        if(error != hipSuccess) {
            cerr << "Unable to copy Temps to device memory" << endl;
            exit(1);
        }
    }
    cout << "Read " << num_rows << " X " << num_cols << " X " << num_slices << " temps" << endl;

    //Reads in the conduction codes for each cell of the simulation and parses
    //the array indexs from the codes
    //Unlike, the Fortran version of the program, the conduction direction codes
    //are ignored since the simulation accounts for them internally
    for (int k = 0; k < num_slices; k++) {
        for(int i = 0; i < num_rows; i++) {
            for(int j = 0; j < num_cols; j++) {
                source_file >> temp_str;
                cond_codes[i*num_cols + j] = atoi(temp_str.c_str());
                cond_tc_index[i*num_cols + j] = atoi(temp_str.substr(0*INDEX_WIDTH,INDEX_WIDTH).c_str())-1;
                cond_hp_index[i*num_cols + j] = atoi(temp_str.substr(1*INDEX_WIDTH,INDEX_WIDTH).c_str())-1;
                use_cond[i*num_cols + j] = atoi(temp_str.substr(2*INDEX_WIDTH,1).c_str());
            }
        }
        //Copies the current conduction code slice to device memory
        error = hipMemcpy(&dev_cond_codes[k*num_rows*num_cols],cond_codes,num_rows*num_cols*sizeof(int),hipMemcpyHostToDevice);
        error = hipMemcpy(&dev_cond_tc_index[k*num_rows*num_cols],cond_tc_index,num_rows*num_cols*sizeof(int),hipMemcpyHostToDevice);
        error = hipMemcpy(&dev_cond_hp_index[k*num_rows*num_cols],cond_hp_index,num_rows*num_cols*sizeof(int),hipMemcpyHostToDevice);
        error = hipMemcpy(&dev_use_cond[k*num_rows*num_cols],use_cond,num_rows*num_cols*sizeof(int),hipMemcpyHostToDevice);
        if(error != hipSuccess) {
            cerr << "Unable to copy conduction codes to device memory" << endl;
            exit(1);
        }
    }
    cout << "Read " << num_rows << " X " << num_cols << " X " << num_slices << " conduction codes" << endl;
    
    //If convection is used for the user specified input file, memory is allocated for its
    //variables and they are read in from the input file
    if(using_convection) {     
        //Allocates memory for the convection variables based on the previously read in simulation
        //parameters
        conv_codes = new int[num_rows*num_cols];
        conv_min_temp_index = new int[num_rows*num_cols];
        conv_direction = new int[num_rows*num_cols];
        conv_vel_index = new int[num_rows*num_cols];
        conv_fluid_index = new int[num_rows*num_cols];
        conv_rock_index = new int[num_rows*num_cols];
        
        //Allocates convection specifc variables in device memory
        error = hipMalloc((void **) &dev_conv_codes,num_cols*num_rows*num_slices*sizeof(int));
        error = hipMalloc((void **) &dev_conv_min_temp_index,num_cols*num_rows*num_slices*sizeof(int));
        error = hipMalloc((void **) &dev_conv_direction,num_cols*num_rows*num_slices*sizeof(int));
        error = hipMalloc((void **) &dev_conv_vel_index,num_cols*num_rows*num_slices*sizeof(int));
        error = hipMalloc((void **) &dev_conv_fluid_index,num_cols*num_rows*num_slices*sizeof(int));
        error = hipMalloc((void **) &dev_conv_rock_index,num_cols*num_rows*num_slices*sizeof(int));
        if(error != hipSuccess) {
            cerr << "Unable to allocate device memory for convection" << endl;
            exit(1);
        }
        
        //Reads in the convection codes for each cell of the simulation and parses the array
        //indexs from the ocdes
        for (int k = 0; k < num_slices; k++) {
            for(int i = 0; i < num_rows; i++) {
                for(int j = 0; j < num_cols; j++) {
                    source_file >> temp_str;
                    conv_codes[i*num_cols + j] = atoi(temp_str.c_str());
                    conv_min_temp_index[i*num_cols + j] = atoi(temp_str.substr(0*INDEX_WIDTH,INDEX_WIDTH).c_str())-1;
                    conv_vel_index[i*num_cols + j] = atoi(temp_str.substr(1*INDEX_WIDTH,INDEX_WIDTH).c_str())-1;
                    conv_fluid_index[i*num_cols + j] = atoi(temp_str.substr(2*INDEX_WIDTH,INDEX_WIDTH).c_str())-1;
                    conv_rock_index[i*num_cols + j] = atoi(temp_str.substr(3*INDEX_WIDTH,INDEX_WIDTH).c_str())-1;
                    conv_direction[i*num_cols + j] = atoi(temp_str.substr(4*INDEX_WIDTH,2).c_str());
                }
            }
            //Copies the current convection code slice to device memory
            error = hipMemcpy(&dev_conv_codes[k*num_cols*num_rows],conv_codes,num_rows*num_cols*sizeof(int),hipMemcpyHostToDevice);
            error = hipMemcpy(&dev_conv_min_temp_index[k*num_cols*num_rows],conv_min_temp_index,num_rows*num_cols*sizeof(int),hipMemcpyHostToDevice);
            error = hipMemcpy(&dev_conv_direction[k*num_cols*num_rows],conv_direction,num_rows*num_cols*sizeof(int),hipMemcpyHostToDevice);
            error = hipMemcpy(&dev_conv_vel_index[k*num_cols*num_rows],conv_vel_index,num_rows*num_cols*sizeof(int),hipMemcpyHostToDevice);
            error = hipMemcpy(&dev_conv_fluid_index[k*num_cols*num_rows],conv_fluid_index,num_rows*num_cols*sizeof(int),hipMemcpyHostToDevice);
            error = hipMemcpy(&dev_conv_rock_index[k*num_cols*num_rows],conv_rock_index,num_rows*num_cols*sizeof(int),hipMemcpyHostToDevice);
            if(error != hipSuccess) {
                cerr << "Unable to copy convection codes to device memory" << endl;
                exit(1);
            }
        }
        cout << "Read " << num_rows << " X " << num_cols << " X " << num_slices << " convection codes" << endl;
    }
    
    //Reads in the Y (column) dimensions and finds the minimum column distance
    for(int i = 0; i < num_cols; i++) {
        source_file >> dim_x[i];
        if(i == 0) {
            min_col_dim = dim_x[0];
            dist_x[0] = dim_x[0]/2.0;
        }
        else {
            if(dim_x[i] < min_col_dim) {
                min_col_dim = dim_x[i];
            }
            dist_x[i] = dist_x[i-1] + dim_x[i-1]/2.0 + dim_x[i]/2.0;
        }
    }
    max_dist_x = dist_x[num_cols-1] + dim_x[num_cols-1]/2.0;
    
    //Copies the x dimensions and distances to device memory
    error = hipMemcpy(dev_dim_x,dim_x,num_cols*sizeof(REAL),hipMemcpyHostToDevice);
    if(error != hipSuccess) {
        cerr << "Unable to copy x dimensions to device" << endl;
        exit(1);
    }
    error = hipMemcpy(dev_dist_x,dist_x,num_cols*sizeof(REAL),hipMemcpyHostToDevice);
    if(error != hipSuccess) {
        cerr << "Unable to copy x dimensions to device" << endl;
        exit(1);
    }
    
    //Reads in the X (row) dimensions and finds the minimum row distance
    for(int i = 0; i < num_rows; i++) {
        source_file >> dim_y[i];
        if(i == 0) {
            min_row_dim = dim_y[i];
            dist_y[0] = dim_y[0]/2.0;
        }
        else {
            if(dim_y[i] < min_row_dim) {
                min_row_dim = dim_y[i];
            }
            dist_y[i] = dist_y[i-1] + dim_y[i-1]/2.0 + dim_y[i]/2.0;
        }
    }
    max_dist_y = dist_y[num_rows-1] + dim_y[num_rows-1]/2.0;
    
    //Copies the y dimensions and distances to device memory
    error = hipMemcpy(dev_dim_y,dim_y,num_rows*sizeof(REAL),hipMemcpyHostToDevice);
    if(error != hipSuccess) {
        cerr << "Unable to copy y dimensions to device" << endl;
        exit(1);
    }
    error = hipMemcpy(dev_dist_y,dist_y,num_rows*sizeof(REAL),hipMemcpyHostToDevice);
    if(error != hipSuccess) {
        cerr << "Unable to copy y dimensions to device" << endl;
        exit(1);
    }
    
    //Reads in the Z (slice depth) dimension and finds the minimum row distance
    for (int i = 0; i < num_slices; i++) {
        source_file >> dim_z[i];
        if (i == 0) {
            min_slice_dim = dim_z[i];
            dist_z[0] = dim_z[0]/2.0;
        }
        else {
            if (dim_z[i] < min_slice_dim) {
                min_slice_dim = dim_z[i];
            }
            dist_z[i] = dist_z[i-1] + dim_z[i-1]/2.0 + dim_z[i]/2.0;
        }
    }
    max_dist_z = dist_z[num_slices-1] + dim_z[num_slices-1]/2.0;
    
    //Copies the z dimensions and distances to device memory
    error = hipMemcpy(dev_dim_z,dim_z,num_slices*sizeof(REAL),hipMemcpyHostToDevice);
    if(error != hipSuccess) {
        cerr << "Unable to copy y dimensions to device" << endl;
        exit(1);
    }
    error = hipMemcpy(dev_dist_z,dist_z,num_slices*sizeof(REAL),hipMemcpyHostToDevice);
    if(error != hipSuccess) {
        cerr << "Unable to copy y dimensions to device" << endl;
        exit(1);
    }
    
    //Reads in the conduction heat production values
    source_file >> num_hp;
    heat_production_values = new REAL[num_hp];
    for(int i = 0; i < num_hp; i++) {
        source_file >> heat_production_values[i];
        heat_production_values[i] /= 1E6;
    }
    
    //Allocates and copies the heat production values to device memory
    error = hipMalloc((void **) &dev_heat_production_values,num_hp*sizeof(REAL));
    error = hipMemcpy(dev_heat_production_values,heat_production_values,num_hp*sizeof(REAL),hipMemcpyHostToDevice);
    if(error != hipSuccess) {
        cerr << "Unable to copy heat production values to device" << endl;
        exit(1);
    }
    cout << "Read "<< num_hp << " heat production values" << endl;
    
    //Reads in the thermal conduction difference values
    //Finds the minimum and maximum thermal conductivity differences and
    //performs some scaling of the conduction associated variables
    source_file >> num_tcd;
    thermal_conduct_diff = new REAL[num_tcd];
    cout << "Converted " << num_tcd << " Thermal Conductivities to Diff. in m^2/y" << endl;
    for(int i = 0; i < num_tcd; i++) {
        source_file >> thermal_conduct_diff[i];
        thermal_conduct_diff[i] *= 14.33;
        if(i == 0) {
            max_thermal_conduct_diff = thermal_conduct_diff[0];
            min_thermal_conduct_diff = thermal_conduct_diff[0];
        }
        else {
            if(thermal_conduct_diff[i] > max_thermal_conduct_diff) {
                max_thermal_conduct_diff = thermal_conduct_diff[i];
            }
            if(thermal_conduct_diff[i] < min_thermal_conduct_diff) {
                min_thermal_conduct_diff = thermal_conduct_diff[i];
            }
        }
        cout << "  " << thermal_conduct_diff[i];
    }
    
    //Allocates and copies the thermal conductivity difference values to device memory
    error = hipMalloc((void **) &dev_thermal_conduct_diff,num_tcd*sizeof(REAL));
    error = hipMemcpy(dev_thermal_conduct_diff,thermal_conduct_diff,num_tcd*sizeof(REAL),hipMemcpyHostToDevice);
    if(error != hipSuccess) {
        cerr << "Unable to copy thermal conductivities to device" << endl;
        exit(1);
    }
    
    //Reads in the convection specific variables if convection
    //is used by the user specified input file
    if(using_convection) {
        //Reads in the fluid heat capacity values
        source_file >> num_hcf;
        heat_capac_fluid = new REAL[num_hcf];
        for(int i = 0; i < num_hcf; i++) {
            source_file >> heat_capac_fluid[i];
        }
        
        //Allocates and copies the fluid heat capacity values to device memory
        error = hipMalloc((void **) &dev_heat_capac_fluid,num_hcf*sizeof(REAL));
        error = hipMemcpy(dev_heat_capac_fluid,heat_capac_fluid,num_hcf*sizeof(REAL),hipMemcpyHostToDevice);
        if(error != hipSuccess) {
            cerr << "Unable to copy fluid heat capacity to device" << endl;
            exit(1);
        }
        
        //Reads in the rock heat capacity values
        source_file >> num_hcr;
        heat_capac_rock = new REAL[num_hcr];
        for(int i = 0; i < num_hcr; i++) {
            source_file >> heat_capac_rock[i];
        }
        
        //Allocates and copies the rock heat capacity values to device memory
        error = hipMalloc((void **) &dev_heat_capac_rock,num_hcr*sizeof(REAL));
        error = hipMemcpy(dev_heat_capac_rock,heat_capac_rock,num_hcr*sizeof(REAL),hipMemcpyHostToDevice);
        if(error != hipSuccess) {
            cerr << "Unable to copy rock heat capacity to device" << endl;
            exit(1);
        }
        
        //Reads in the minimum convection temperatures
        source_file >> num_mtc;
        min_temp_conv = new REAL[num_mtc];
        for(int i = 0; i < num_mtc; i++) {
            
            source_file >> min_temp_conv[i];
        }
        
        //Allocates and copies the minimum convection temperature values to device memory
        error = hipMalloc((void **) &dev_min_temp_conv,num_mtc*sizeof(REAL));
        error = hipMemcpy(dev_min_temp_conv,min_temp_conv,num_mtc*sizeof(REAL),hipMemcpyHostToDevice);
        if(error != hipSuccess) {
            cerr << "Unable to copy minimum temp for convection to device" << endl;
            exit(1);
        }
        
        //Reads in the convection velocities
        source_file >> num_vel;
        vel = new REAL[num_vel];
        for(int i = 0; i < num_vel; i++) {
            source_file >> vel[i];
        }
        cout << endl << "Read " << num_vel << " Velocities in m/yr" << endl;
        
        //Allocates and copies the convection velocities to device memory
        error = hipMalloc((void **) &dev_vel,num_vel*sizeof(REAL));
        error = hipMemcpy(dev_vel,vel,num_vel*sizeof(REAL),hipMemcpyHostToDevice);
        if(error != hipSuccess) {
            cerr << "Unable to copy convection velocities to device" << endl;
            exit(1);
        }
        
        //Finds the maximum convection velocity
        max_vel = vel[0];
        cout << " " << vel[0];
        for(int i = 1 ; i < num_vel; i++) {
            if(vel[i] > max_vel) {
                max_vel = vel[i];
            }
            cout << " " << vel[i];
        }
        cout << endl;
    }
    
    //Closes the input file
    source_file.close();
    
    /*
    T1 = max_thermal_conduct_diff;
    T2 = min_col_dim;
    T3 = min_row_dim;
    */
    //Finds the convection time increment
    if(using_convection) {
        if(min_col_dim > min_row_dim) {
            tic = min_row_dim/max_vel;
        }
        else {
            tic = min_col_dim/max_vel;
        }
    }
    //Calculates the maximum time step of the simulation
    if(min_col_dim < min_row_dim) {
        time_step = min_col_dim*min_col_dim/(5*max_thermal_conduct_diff);
    }
    else {
        time_step = min_row_dim*min_row_dim/(5*max_thermal_conduct_diff);
    }
    cout << endl << "Done Loading Input File" << endl;
}

/**
 * Saves the current state of the simulation, using the same format
 * as the input file
 */
void save_model_state() {
    ofstream output_file;    //Output file stream
    
    //Opens the output file for writing
    output_file.open(output_filename.c_str(),ios::out);
    if(!output_file.is_open()) {
        cerr << "Failed to write state to file" << endl;
        exit(1);
    }
    else {
        //Prints the simulation parameters to the output file
        output_file << setw(20) << num_rows << " " << setw(20) << num_cols << " " << setw(20) << num_slices << setw(20) << using_convection << endl;
        output_file << setw(20) << fixed << setprecision(OUT_PRECISION) << chf*1000.0 << " " << setw(20) << initial_time + sim_time << endl;
        output_file << title << endl;
        
        output_file << setprecision(OUT_PRECISION);
        //Prints the current temperature array of the simulation
        for (int k = 0; k < num_slices; k++) {
            //Copies the current temperature slice into host memory
            error = hipMemcpy(temp,&dev_temp[k*num_rows*num_cols],num_rows*num_cols*sizeof(REAL),hipMemcpyDeviceToHost);
            if(error != hipSuccess) {
                cerr << "Unable to copy convection velocities to device" << endl;
                exit(1);
            }
            
            for(int i = 0; i < num_rows; i++) {
                for(int j = 0; j < num_cols; j++) {
                    output_file << " " << setw(OUT_PRECISION+5) << temp[i*num_cols + j];
                }
                output_file << endl;
            }
            output_file << endl;
        }
        
        //Prints the conduction codes of the simulation to the output file
        output_file << setfill('0');
        for (int k = 0; k < num_slices; k++) {
            //Copies the current conduction code slice into host memory
            error = hipMemcpy(cond_codes,&dev_cond_codes[k*num_rows*num_cols],num_rows*num_cols*sizeof(int),hipMemcpyDeviceToHost);
            if(error != hipSuccess) {
                cerr << "Unable to copy convection velocities to device" << endl;
                exit(1);
            }
            
            for(int i = 0; i < num_rows; i++) {
                for(int j = 0; j < num_cols; j++) {
                    output_file << " " << setw(2*INDEX_WIDTH+1) << cond_codes[i*num_cols + j];
                }
                output_file << endl;
            }
            output_file << endl;
        }
        
        //Prints the convection codes to the output file if convection is being used
        if(using_convection) {
            for (int k = 0; k < num_slices; k++) {
                //Copies the current convection code slice into host memory
                error = hipMemcpy(conv_codes,&dev_conv_codes[k*num_rows*num_cols],num_rows*num_cols*sizeof(int),hipMemcpyDeviceToHost);
                if(error != hipSuccess) {
                    cerr << "Unable to copy convection velocities to device" << endl;
                    exit(1);
                }
                
                for(int i = 0; i < num_rows; i++) {
                    for(int j = 0; j < num_cols; j++) {
                        output_file << " " << setw(4*INDEX_WIDTH+2) << conv_codes[i*num_cols + j];
                    }
                    output_file << endl;
                }
                output_file << endl;
            }
        }
        
        output_file << setfill(' ');
        output_file << setprecision(3);
        //Prints the column (X) dimensions of the simulation to the output file
        for(int i = 0; i < num_cols; i++) {
            output_file << " " << dim_x[i];
        }
        output_file << endl;
        
        //Prints the row (Y) dimensions of the simulation to the output file
        for(int i = 0; i < num_rows; i++) {
            output_file << " " << dim_y[i];
        }
        output_file << endl;
        
        // Prints the slice (Z) dimensions of the simulation to the oputput file

        for (int i = 0; i < num_slices; i++) {
            output_file << " " << dim_z[i];
        }
        output_file << endl;

        //Prints the heat production values of the simulation to the output file
        output_file << " " << num_hp;
        for(int i = 0; i < num_hp; i++) {
            output_file << " " << scientific << heat_production_values[i]*1E6;
        }
        output_file << endl;
        
        //Prints the thermal conductivity difference values to the output file
        output_file << " " << num_tcd;
        for(int i = 0; i < num_tcd; i++) {
            output_file << " " << thermal_conduct_diff[i]/14.33;
        }
        output_file << endl;
        
        //Prints the convection specific variables to the output file if convection is used
        if(using_convection) {
            //Prints the fluid heat capacity values to the output file
            output_file << " " << num_hcf;
            for(int i = 0; i < num_hcf; i++) {
                output_file << " " << heat_capac_fluid[i];
            }
            output_file << endl;
            
            //Prints the rock heat capacity values to the output file
            output_file << " " << num_hcr;
            for(int i = 0; i < num_hcr; i++) {
                output_file << " " << heat_capac_rock[i];
            }
            output_file << endl;
            
            //Prints the minimum convection temps to the output file
            output_file << " " << num_mtc;
            for(int i = 0; i < num_mtc; i++) {
                output_file << " " << min_temp_conv[i];
            }
            output_file << endl;
            
            //Prints the convection velocities to the output file
            output_file << " " << num_vel;
            for(int i = 0; i < num_vel; i++) {
                output_file << " " << vel[i];
            }
            output_file << endl;
        }
        
        //Closes the output file
        output_file.close();
    }
}

/**
 * Saves the current temperatures of the simulation to a DSAA surfer grid file
 */
void save_surfer() {
    ofstream output_file;            //Output file stream
    ostringstream oss;
    string filename, extension;
    filename = output_su_filename.substr(0,output_su_filename.length()-4);
    extension = output_su_filename.substr(output_su_filename.length()-4,4);
    
    for(int k = 0; k < num_slices; k++) {
        oss.str("");
        oss.clear();
        oss << filename << setfill('0') << setw(su_num_width) << k << extension;
        //Opens the output file for writting
        output_file.open(oss.str().c_str(),ios::out);
        if(!output_file.is_open()) {
            cerr << "Failed to write surfer file" << endl;
            exit(1);
        }
        else {
            REAL min_temp, max_temp, temp_range;    //Minimum and maximum temperatures.
            REAL xmax,ymin;                         //Maximum x and minimum y distances
            
            //Copies the current temperature slice to host memory
            error = hipMemcpy(temp,&dev_temp[k*num_rows*num_cols],num_rows*num_cols*sizeof(REAL),hipMemcpyDeviceToHost);
            if(error != hipSuccess) {
                cerr << "Unable to copy convection velocities to device" << endl;
                exit(1);
            }
            
            //Finds the minimum and maximum temps in the temperature array
            min_temp = max_temp = temp[0];
            for(int i = 0; i < num_rows; i++) {
                for(int j = 0; j < num_cols; j++) {
                    if(temp[i*num_cols + j] > max_temp) {
                        max_temp = temp[i*num_cols + j];
                    }
                    if(temp[i*num_cols + j] < min_temp) {
                        min_temp = temp[i*num_cols + j];
                    }
                }
            }
            
            //Calculates the temperature range.
            temp_range = max_temp - min_temp;
            if(temp_range == 0) {
                temp_range = 1.0;
            }
            
            //Calculates the maximum x distance and the
            //minimum y distance
            xmax = dim_x[0]*num_cols;
            ymin = dim_y[0]*num_rows;
            if(dim_x[0] < 0.01) {
                xmax *= 1000;
            }
            else if(dim_x[0] < 0.1) {
                xmax *= 100;
            }
            else if(dim_x[0] < 1) {
                xmax *= 10;
            }
            if(dim_y[0] < 0.01) {
                ymin *= 1000;
            }
            else if(dim_y[0] < 0.1) {
                ymin *= 100;
            }
            else if(dim_y[0] < 1) {
                ymin *= 10;
            }
            
            //Prints the DSAA surfer grid parameters to the output file
            output_file << "DSAA" << endl;
            output_file << setw(20) << num_cols << " " << setw(20) << num_rows << endl;
            output_file << fixed << setprecision(3) << setw(20) << 0.0 << " " << setw(20) << xmax << endl;
            output_file << setw(20) << -ymin << " " << setw(20) << 0.0 << endl;
            output_file << setw(20) << setprecision(OUT_PRECISION) << min_temp << " " << setw(20) << max_temp << endl;
            
            //Prints the temperature array to the output file
            for(int i = num_rows-1; i >= 0; i--) {
                for(int j = 0; j < num_cols; j++) {
                    output_file << " " << setw(OUT_PRECISION+5) << temp[i*num_cols + j];
                }
                output_file << endl;
            }
            
            //Closes the output file
            output_file.close();
        }
    }
}

/**
 * Calculates and returns the heat flow per year between two cells in the X direction
 * based on the provided indexes
 */
__device__ REAL cond_add_x(int row1, int col1, int slice1, int row2, int col2, int slice2, int num_rows, int num_cols, int num_slices, REAL *dim_x, REAL *dim_y, REAL *dim_z, REAL *temp, REAL *next_temp, REAL *thermal_conduct_diff, int *cond_tc_index) {
    REAL temp_diff;    //Temperature difference between the two cells
    REAL ad;           //
    
    temp_diff = temp[slice1*num_rows*num_cols + row1*num_cols + col2] - temp[slice1*num_rows*num_cols + row1*num_cols + col1];
    ad = dim_x[col2]/thermal_conduct_diff[cond_tc_index[slice1*num_rows*num_cols + row1*num_cols + col2]] + dim_x[col1]/thermal_conduct_diff[cond_tc_index[slice1*num_rows*num_cols + row1*num_cols + col1]];

    return 2*temp_diff/(ad*dim_x[col1]);
}

/**
 * Calculates and returns the heat flow per year between two cells in the Y direction
 * based on the provided indexes
 */
__device__ REAL cond_add_y(int row1, int col1, int slice1, int row2, int col2, int slice2, int num_rows, int num_cols, int num_slices, REAL *dim_x, REAL *dim_y, REAL *dim_z, REAL *temp, REAL *next_temp, REAL *thermal_conduct_diff, int *cond_tc_index) {
    REAL temp_diff;    //Temperature difference between the two cells
    REAL ad;           //
    
    temp_diff = temp[slice1*num_rows*num_cols + row2*num_cols + col1] - temp[slice1*num_rows*num_cols + row1*num_cols + col1];
    ad = dim_y[row2]/thermal_conduct_diff[cond_tc_index[slice1*num_rows*num_cols + row2*num_cols + col1]] + dim_y[row1]/thermal_conduct_diff[cond_tc_index[slice1*num_rows*num_cols + row1*num_cols + col1]];

    return 2*temp_diff/(ad*dim_y[row1]);
}

/**
 * Calculates and returns the heat flow per year between two cells in the Z direction
 * based on the provided indexes
 */
__device__ REAL cond_add_z(int row1, int col1, int slice1, int row2, int col2, int slice2, int num_rows, int num_cols, int num_slices, REAL *dim_x, REAL *dim_y, REAL *dim_z, REAL *temp, REAL *next_temp, REAL *thermal_conduct_diff, int *cond_tc_index) {
    if(num_slices == 1) {
        return 0.0;
    }
    REAL temp_diff;
    REAL ad;
    temp_diff = temp[slice2*num_rows*num_cols + row1*num_cols + col1] - temp[slice1*num_rows*num_cols + row1*num_cols + col1];
    ad = dim_z[slice2]/thermal_conduct_diff[cond_tc_index[slice2*num_rows*num_cols + row1*num_cols + col1]] + dim_z[slice1]/thermal_conduct_diff[cond_tc_index[slice1*num_rows*num_cols + row1*num_cols + col1]];

    return 2*temp_diff/(ad*dim_z[slice1]);
}

/**
 * Calculates the in-plane heat flow due to conduction in a given slice k.
 * If slices == 1
 *   2d simulation, return 0 for 3rd dimension heat transfer
 * else
 *   Calculate and return heat flow per year between two cells in the Z direction
 */
__device__ REAL in_plane_cond(int i, int j, int k, int num_rows, int num_cols, int num_slices, REAL *dim_x, REAL *dim_y, REAL *dim_z, REAL DHF, REAL *temp, REAL *next_temp, REAL *thermal_conduct_diff, int *cond_tc_index) {
    REAL heat_flow_x;
    REAL heat_flow_y;

    /* k is fixed */
    if(i == 0 && j == 0) { //Top left corner of slice
        heat_flow_x = cond_add_x(i,j,k,i,j+1,k,num_rows,num_cols,num_slices,dim_x,dim_y,dim_z,temp,next_temp,thermal_conduct_diff,cond_tc_index);
        heat_flow_y = cond_add_y(i,j,k,i+1,j,k,num_rows,num_cols,num_slices,dim_x,dim_y,dim_z,temp,next_temp,thermal_conduct_diff,cond_tc_index);
        next_temp[k*num_rows*num_cols + i*num_cols + j] = 0.0;
    }
    else if(i == 0 && j == num_cols-1) { //Top right corner of slice
        heat_flow_x = cond_add_x(i,j,k,i,j-1,k,num_rows,num_cols,num_slices,dim_x,dim_y,dim_z,temp,next_temp,thermal_conduct_diff,cond_tc_index);
        heat_flow_y = cond_add_y(i,j,k,i+1,j,k,num_rows,num_cols,num_slices,dim_x,dim_y,dim_z,temp,next_temp,thermal_conduct_diff,cond_tc_index);
        next_temp[k*num_rows*num_cols + i*num_cols + j] = 0.0;
    }
    else if(i == 0) { //Top of slice
        heat_flow_x = cond_add_x(i,j,k,i,j+1,k,num_rows,num_cols,num_slices,dim_x,dim_y,dim_z,temp,next_temp,thermal_conduct_diff,cond_tc_index) + cond_add_x(i,j,k,i,j-1,k,num_rows,num_cols,num_slices,dim_x,dim_y,dim_z,temp,next_temp,thermal_conduct_diff,cond_tc_index);
        heat_flow_y = cond_add_y(i,j,k,i+1,j,k,num_rows,num_cols,num_slices,dim_x,dim_y,dim_z,temp,next_temp,thermal_conduct_diff,cond_tc_index);
        next_temp[k*num_rows*num_cols + i*num_cols + j] = 0.0;
    }
    else if(i == num_rows-1 && j == 0) { //Bottom left corner of slice
        heat_flow_x = cond_add_x(i,j,k,i,j+1,k,num_rows,num_cols,num_slices,dim_x,dim_y,dim_z,temp,next_temp,thermal_conduct_diff,cond_tc_index);
        heat_flow_y = cond_add_y(i,j,k,i-1,j,k,num_rows,num_cols,num_slices,dim_x,dim_y,dim_z,temp,next_temp,thermal_conduct_diff,cond_tc_index);
        next_temp[k*num_rows*num_cols + i*num_cols + j] = DHF/dim_y[i];    //Constant heat flow at the bottom of the model
    }
    else if(i == num_rows-1 && j == num_cols-1) { //Bottom right corner of slice
        heat_flow_x = cond_add_x(i,j,k,i,j-1,k,num_rows,num_cols,num_slices,dim_x,dim_y,dim_z,temp,next_temp,thermal_conduct_diff,cond_tc_index);
        heat_flow_y = cond_add_y(i,j,k,i-1,j,k,num_rows,num_cols,num_slices,dim_x,dim_y,dim_z,temp,next_temp,thermal_conduct_diff,cond_tc_index);
        next_temp[k*num_rows*num_cols + i*num_cols + j] = DHF/dim_y[i];    //Constant heat flow at the bottom of the model
    }
    else if(i == num_rows-1) { //Bottom
        heat_flow_x = cond_add_x(i,j,k,i,j+1,k,num_rows,num_cols,num_slices,dim_x,dim_y,dim_z,temp,next_temp,thermal_conduct_diff,cond_tc_index) + cond_add_x(i,j,k,i,j-1,k,num_rows,num_cols,num_slices,dim_x,dim_y,dim_z,temp,next_temp,thermal_conduct_diff,cond_tc_index);
        heat_flow_y = cond_add_y(i,j,k,i-1,j,k,num_rows,num_cols,num_slices,dim_x,dim_y,dim_z,temp,next_temp,thermal_conduct_diff,cond_tc_index);
        next_temp[k*num_rows*num_cols + i*num_cols + j] = DHF/dim_y[i];    //Constant heat flow at the bottom of the model
    }
    else if(j == 0) { //Left side of slice
        heat_flow_x = cond_add_x(i,j,k,i,j+1,k,num_rows,num_cols,num_slices,dim_x,dim_y,dim_z,temp,next_temp,thermal_conduct_diff,cond_tc_index);
        heat_flow_y = cond_add_y(i,j,k,i-1,j,k,num_rows,num_cols,num_slices,dim_x,dim_y,dim_z,temp,next_temp,thermal_conduct_diff,cond_tc_index) + cond_add_y(i,j,k,i+1,j,k,num_rows,num_cols,num_slices,dim_x,dim_y,dim_z,temp,next_temp,thermal_conduct_diff,cond_tc_index);
        next_temp[k*num_rows*num_cols + i*num_cols + j] = 0.0;
    }
    else if(j == num_cols-1) { //Right side of slice
        heat_flow_x = cond_add_x(i,j,k,i,j-1,k,num_rows,num_cols,num_slices,dim_x,dim_y,dim_z,temp,next_temp,thermal_conduct_diff,cond_tc_index);
        heat_flow_y = cond_add_y(i,j,k,i-1,j,k,num_rows,num_cols,num_slices,dim_x,dim_y,dim_z,temp,next_temp,thermal_conduct_diff,cond_tc_index) + cond_add_y(i,j,k,i+1,j,k,num_rows,num_cols,num_slices,dim_x,dim_y,dim_z,temp,next_temp,thermal_conduct_diff,cond_tc_index);
        next_temp[k*num_rows*num_cols + i*num_cols + j] = 0.0;
    }
    else { //Middle of slice
        heat_flow_x = cond_add_x(i,j,k,i,j-1,k,num_rows,num_cols,num_slices,dim_x,dim_y,dim_z,temp,next_temp,thermal_conduct_diff,cond_tc_index) + cond_add_x(i,j,k,i,j+1,k,num_rows,num_cols,num_slices,dim_x,dim_y,dim_z,temp,next_temp,thermal_conduct_diff,cond_tc_index);
        heat_flow_y = cond_add_y(i,j,k,i-1,j,k,num_rows,num_cols,num_slices,dim_x,dim_y,dim_z,temp,next_temp,thermal_conduct_diff,cond_tc_index) + cond_add_y(i,j,k,i+1,j,k,num_rows,num_cols,num_slices,dim_x,dim_y,dim_z,temp,next_temp,thermal_conduct_diff,cond_tc_index);
        next_temp[k*num_rows*num_cols + i*num_cols + j] = 0.0;
    }
    return (heat_flow_x + heat_flow_y);
}

/**
 * Conduction Kernel
 * Updates the temperature array using 3D conduction with finite
 * difference heat flow.
 */
__global__ void conduction_kernel(int num_cells, int num_rows, int num_cols, int num_slices, REAL *dim_x, REAL *dim_y, REAL *dim_z, REAL DHF, REAL time_step, REAL *temp, REAL *next_temp, int *use_cond, REAL *heat_production_values, int *cond_hp_index, REAL *thermal_conduct_diff, int *cond_tc_index){
    unsigned long long id = blockIdx.x*blockDim.x+threadIdx.x;		//Thread ID
    if(id < num_cells) {
        int k = id/(num_rows*num_cols);
        int i = (id- k*num_rows*num_cols)/num_cols;
        int j = id - k*num_rows*num_cols - i*num_cols;
        if(use_cond[k*num_rows*num_cols + i*num_cols + j] == 1) {
            REAL heatflow_in_plane;    //Heat flow occuring inside of plane
            REAL heatflow_cross_plane; //Heat flow into and out of plane/slice
            if (k == 0) { // First slice
                heatflow_in_plane = in_plane_cond(i,j,k,num_rows,num_cols,num_slices,dim_x,dim_y,dim_z,DHF,temp,next_temp,thermal_conduct_diff,cond_tc_index);         // heat transfer inside of plane
                heatflow_cross_plane = cond_add_z(i,j,k,i,j,k+1,num_rows,num_cols,num_slices,dim_x,dim_y,dim_z,temp,next_temp,thermal_conduct_diff,cond_tc_index); // slice-to-slice heat transfer.  first slice, so only from next slice transfers heat.
            }
            else if (k == num_slices - 1) {   // Last slice
                heatflow_in_plane = in_plane_cond(i,j,k,num_rows,num_cols,num_slices,dim_x,dim_y,dim_z,DHF,temp,next_temp,thermal_conduct_diff,cond_tc_index);         // heat transfer inside of plane
                heatflow_cross_plane = cond_add_z(i,j,k,i,j,k-1,num_rows,num_cols,num_slices,dim_x,dim_y,dim_z,temp,next_temp,thermal_conduct_diff,cond_tc_index); // slice-to-slice heat transfer.  last slice, so only previous slice transfers heat.
            }
            else {  // Middle
                heatflow_in_plane = in_plane_cond(i,j,k,num_rows,num_cols,num_slices,dim_x,dim_y,dim_z,DHF,temp,next_temp,thermal_conduct_diff,cond_tc_index);                                     // you get the idea
                heatflow_cross_plane = cond_add_z(i,j,k,i,j,k+1,num_rows,num_cols,num_slices,dim_x,dim_y,dim_z,temp,next_temp,thermal_conduct_diff,cond_tc_index) + cond_add_z(i,j,k,i,j,k-1,num_rows,num_cols,num_slices,dim_x,dim_y,dim_z,temp,next_temp,thermal_conduct_diff,cond_tc_index); // slice-to-slice heat transfer. Middle, so both next and previous.
            }
            //Heat flow from the adjacent cells
            next_temp[k*num_rows*num_cols + i*num_cols + j] += temp[k*num_rows*num_cols + i*num_cols + j] + time_step*(heatflow_in_plane + heatflow_cross_plane);
            //Heat flow due to radioactive heat production
            next_temp[k*num_rows*num_cols + i*num_cols + j] += heat_production_values[cond_hp_index[k*num_rows*num_cols + i*num_cols + j]]*time_step/DTC;
        }
        else {
            next_temp[k*num_rows*num_cols + i*num_cols + j] = temp[k*num_rows*num_cols + i*num_cols + j];
        }
    }
}


/**
 * Wrapper Function for the conduction kernel
 */
void conduction_cuda() {

    //Calls the conduction kernel
    conduction_kernel<<<dimGrid,dimBlock>>>(num_cells,num_rows,num_cols,num_slices,dev_dim_x,dev_dim_y,dev_dim_z,DHF,time_step,dev_temp,dev_next_temp,dev_use_cond,dev_heat_production_values,dev_cond_hp_index,dev_thermal_conduct_diff,dev_cond_tc_index);
    
    //Waits for the kernel to finish executing
    hipDeviceSynchronize();
    
    //Checks if an error occured during execution of the kernel
    error = hipGetLastError();
    if(error != hipSuccess) {
        cerr << "Error while executing conduction kernel" << endl;
        cerr << error << " : " <<  hipGetErrorString(error) << endl;
        cerr << "Exiting the Program" << endl;
        exit(1);
    }
    
    //Swaps the device temperature arrays
    swap_temp_array_cuda();
}

/**
 * Performs convection between two specified cells
 */
__device__ void perform_convection(int row1, int col1, int slice1, int row2, int col2, int slice2, int num_rows, int num_cols, int num_slices, REAL *dist_x, REAL *dist_y, REAL *dist_z, REAL time_inc, REAL *temp, REAL *next_temp, REAL *min_temp_conv, int *conv_min_temp_index, REAL *heat_capac_fluid, int *conv_fluid_index, REAL *heat_capac_rock, int *conv_rock_index, REAL *vel, int *conv_vel_index) {
    REAL avg_x_dim;    //distance between two temperature cells in the x direction
    REAL avg_y_dim;    //distance between two temperature cells in the y direction
    REAL avg_z_dim;    //distance between two temperature cells in the z direction
    REAL amt;          //
    REAL dist;         //Distance between the two cells
    REAL ratio;        //Ratio of amt to distance
    
    //Checks if the specified cell is within the bounds of the simulation and if it has a high enough
    //temperature to perform convection
    if((row2 >= 0) && (row2 < num_rows) && (col2 >= 0) && (col2 < num_cols) && (slice2 >= 0) && (slice2 < num_slices) && (temp[slice2*num_rows*num_cols + row2*num_cols + col2] - min_temp_conv[conv_min_temp_index[slice1*num_rows*num_cols + row1*num_cols + col1]] >= 0)) {
        avg_x_dim = dist_x[col1] - dist_x[col2];
        avg_y_dim = dist_y[row1] - dist_y[row2];
        avg_z_dim = dist_z[slice1] - dist_z[slice2];

        amt = (vel[conv_vel_index[slice1*num_rows*num_cols + row1*num_cols + col1]]*heat_capac_fluid[conv_fluid_index[slice1*num_rows*num_cols + row1*num_cols + col1]]/heat_capac_rock[conv_rock_index[slice1*num_rows*num_cols + row1*num_cols + col1]])*time_inc;
        dist = sqrt(avg_x_dim*avg_x_dim + avg_y_dim*avg_y_dim + avg_z_dim*avg_z_dim);
        ratio = amt/dist;
        if(ratio > 1) {
            ratio = 0.999999;
        }
        next_temp[slice1*num_rows*num_cols + row1*num_cols + col1] = temp[slice1*num_rows*num_cols + row1*num_cols + col1] + ratio *(temp[slice2*num_rows*num_cols + row2*num_cols + col2]-temp[slice1*num_rows*num_cols + row1*num_cols + col1]);
    }
    else {
        next_temp[slice1*num_rows*num_cols + row1*num_cols + col1] = temp[slice1*num_rows*num_cols + row1*num_cols + col1];
    }
}

/**
 * Convection Kernel
 * Updates the temperature array using convection
 */
__global__ void convection_kernel(unsigned long long num_cells, int num_rows, int num_cols, int num_slices, REAL *dist_x, REAL *dist_y, REAL *dist_z, REAL time_inc, REAL *temp, REAL *next_temp, int *conv_codes, int *conv_direction, REAL *min_temp_conv, int *conv_min_temp_index, REAL *heat_capac_fluid, int *conv_fluid_index, REAL *heat_capac_rock, int *conv_rock_index, REAL *vel, int *conv_vel_index) {
    unsigned long long id = blockIdx.x*blockDim.x+threadIdx.x;		//Thread ID
    if(id < num_cells) {
        int k = id/(num_rows*num_cols);
        int i = (id- k*num_rows*num_cols)/num_cols;
        int j = id - k*num_rows*num_cols - i*num_cols;
        
        //Checks if convection can occur for the specified cell
        if((conv_codes[k*num_rows*num_cols + i*num_cols + j] <= 0) || (i == 0) || (conv_direction[k*num_rows*num_cols + i*num_cols + j] == 5) || (conv_direction[k*num_rows*num_cols + i*num_cols + j] < 1) || (conv_direction[k*num_rows*num_cols + i*num_cols + j] > 27)) {
            next_temp[k*num_rows*num_cols + i*num_cols + j] = temp[k*num_rows*num_cols + i*num_cols + j];
        }
        else {
            //Performs convection based on the convection direction code
            switch(conv_direction[k*num_rows*num_cols + i*num_cols + j]) {
                 /**
                  * IN-PLANE convection -- 1 through 9. These codes are for convection taking place in the current, "k-th" plane
                  *     1   2   3
                  *     4   5   6
                  *     7   8   9
                  */
                case 1:
                    perform_convection(i,j,k,i-1,j-1,k,num_rows,num_cols,num_slices,dist_x,dist_y,dist_z,time_inc,temp,next_temp,min_temp_conv,conv_min_temp_index,heat_capac_fluid,conv_fluid_index,heat_capac_rock,conv_rock_index,vel,conv_vel_index);
                    break;
                case 2:                                                           
                    perform_convection(i,j,k,i-1,j,k,num_rows,num_cols,num_slices,dist_x,dist_y,dist_z,time_inc,temp,next_temp,min_temp_conv,conv_min_temp_index,heat_capac_fluid,conv_fluid_index,heat_capac_rock,conv_rock_index,vel,conv_vel_index);                    
                    break;                                                        
                case 3:                                                  
                    perform_convection(i,j,k,i-1,j+1,k,num_rows,num_cols,num_slices,dist_x,dist_y,dist_z,time_inc,temp,next_temp,min_temp_conv,conv_min_temp_index,heat_capac_fluid,conv_fluid_index,heat_capac_rock,conv_rock_index,vel,conv_vel_index);
                    break;
                case 4:
                    perform_convection(i,j,k,i,j-1,k,num_rows,num_cols,num_slices,dist_x,dist_y,dist_z,time_inc,temp,next_temp,min_temp_conv,conv_min_temp_index,heat_capac_fluid,conv_fluid_index,heat_capac_rock,conv_rock_index,vel,conv_vel_index);
                    break;
                case 6:
                    perform_convection(i,j,k,i,j+1,k,num_rows,num_cols,num_slices,dist_x,dist_y,dist_z,time_inc,temp,next_temp,min_temp_conv,conv_min_temp_index,heat_capac_fluid,conv_fluid_index,heat_capac_rock,conv_rock_index,vel,conv_vel_index);
                    break;
                case 7:
                    perform_convection(i,j,k,i+1,j-1,k,num_rows,num_cols,num_slices,dist_x,dist_y,dist_z,time_inc,temp,next_temp,min_temp_conv,conv_min_temp_index,heat_capac_fluid,conv_fluid_index,heat_capac_rock,conv_rock_index,vel,conv_vel_index);
                    break;
                case 8:
                    perform_convection(i,j,k,i+1,j,k,num_rows,num_cols,num_slices,dist_x,dist_y,dist_z,time_inc,temp,next_temp,min_temp_conv,conv_min_temp_index,heat_capac_fluid,conv_fluid_index,heat_capac_rock,conv_rock_index,vel,conv_vel_index);
                    break;
                case 9:
                    perform_convection(i,j,k,i+1,j+1,k,num_rows,num_cols,num_slices,dist_x,dist_y,dist_z,time_inc,temp,next_temp,min_temp_conv,conv_min_temp_index,heat_capac_fluid,conv_fluid_index,heat_capac_rock,conv_rock_index,vel,conv_vel_index);
                    break;
                /**
                 * CROSS-PLANE convection (previous "k-1th" plane) -- 10 through 18
                 *      10  11  12
                 *      13  14  15
                 *      16  17  18      
                 */
                case 10:                                   
                    perform_convection(i,j,k,i-1,j-1,k-1,num_rows,num_cols,num_slices,dist_x,dist_y,dist_z,time_inc,temp,next_temp,min_temp_conv,conv_min_temp_index,heat_capac_fluid,conv_fluid_index,heat_capac_rock,conv_rock_index,vel,conv_vel_index);
                    break;
                case 11:
                    perform_convection(i,j,k,i-1,j,k-1,num_rows,num_cols,num_slices,dist_x,dist_y,dist_z,time_inc,temp,next_temp,min_temp_conv,conv_min_temp_index,heat_capac_fluid,conv_fluid_index,heat_capac_rock,conv_rock_index,vel,conv_vel_index);  
                    break;
                case 12:
                    perform_convection(i,j,k,i-1,j+1,k-1,num_rows,num_cols,num_slices,dist_x,dist_y,dist_z,time_inc,temp,next_temp,min_temp_conv,conv_min_temp_index,heat_capac_fluid,conv_fluid_index,heat_capac_rock,conv_rock_index,vel,conv_vel_index);
                    break;
                case 13:
                    perform_convection(i,j,k,i,j-1,k-1,num_rows,num_cols,num_slices,dist_x,dist_y,dist_z,time_inc,temp,next_temp,min_temp_conv,conv_min_temp_index,heat_capac_fluid,conv_fluid_index,heat_capac_rock,conv_rock_index,vel,conv_vel_index);
                    break;
                case 14:
                    perform_convection(i,j,k,i,j,k-1,num_rows,num_cols,num_slices,dist_x,dist_y,dist_z,time_inc,temp,next_temp,min_temp_conv,conv_min_temp_index,heat_capac_fluid,conv_fluid_index,heat_capac_rock,conv_rock_index,vel,conv_vel_index);
                    break;
                case 15:
                    perform_convection(i,j,k,i,j+1,k-1,num_rows,num_cols,num_slices,dist_x,dist_y,dist_z,time_inc,temp,next_temp,min_temp_conv,conv_min_temp_index,heat_capac_fluid,conv_fluid_index,heat_capac_rock,conv_rock_index,vel,conv_vel_index);
                    break;
                case 16:
                    perform_convection(i,j,k,i+1,j-1,k-1,num_rows,num_cols,num_slices,dist_x,dist_y,dist_z,time_inc,temp,next_temp,min_temp_conv,conv_min_temp_index,heat_capac_fluid,conv_fluid_index,heat_capac_rock,conv_rock_index,vel,conv_vel_index);
                    break;
                case 17:
                    perform_convection(i,j,k,i+1,j,k-1,num_rows,num_cols,num_slices,dist_x,dist_y,dist_z,time_inc,temp,next_temp,min_temp_conv,conv_min_temp_index,heat_capac_fluid,conv_fluid_index,heat_capac_rock,conv_rock_index,vel,conv_vel_index);
                    break;
                case 18:
                    perform_convection(i,j,k,i+1,j+1,k-1,num_rows,num_cols,num_slices,dist_x,dist_y,dist_z,time_inc,temp,next_temp,min_temp_conv,conv_min_temp_index,heat_capac_fluid,conv_fluid_index,heat_capac_rock,conv_rock_index,vel,conv_vel_index);
                    break;
                /**
                 * CROSS-PLANE convection ("k+1th" plane) -- 19 through 27 
                 *      19  20  21
                 *      22  23  24
                 *      25  26  27
                 */
                case 19:                                   
                    perform_convection(i,j,k,i-1,j-1,k-1,num_rows,num_cols,num_slices,dist_x,dist_y,dist_z,time_inc,temp,next_temp,min_temp_conv,conv_min_temp_index,heat_capac_fluid,conv_fluid_index,heat_capac_rock,conv_rock_index,vel,conv_vel_index);
                    break;
                case 20:
                    perform_convection(i,j,k,i-1,j,k-1,num_rows,num_cols,num_slices,dist_x,dist_y,dist_z,time_inc,temp,next_temp,min_temp_conv,conv_min_temp_index,heat_capac_fluid,conv_fluid_index,heat_capac_rock,conv_rock_index,vel,conv_vel_index);
                    break;
                case 21:
                    perform_convection(i,j,k,i-1,j+1,k-1,num_rows,num_cols,num_slices,dist_x,dist_y,dist_z,time_inc,temp,next_temp,min_temp_conv,conv_min_temp_index,heat_capac_fluid,conv_fluid_index,heat_capac_rock,conv_rock_index,vel,conv_vel_index);
                    break;
                case 22:
                    perform_convection(i,j,k,i,j-1,k-1,num_rows,num_cols,num_slices,dist_x,dist_y,dist_z,time_inc,temp,next_temp,min_temp_conv,conv_min_temp_index,heat_capac_fluid,conv_fluid_index,heat_capac_rock,conv_rock_index,vel,conv_vel_index);
                    break;
                case 23:
                    perform_convection(i,j,k,i,j,k-1,num_rows,num_cols,num_slices,dist_x,dist_y,dist_z,time_inc,temp,next_temp,min_temp_conv,conv_min_temp_index,heat_capac_fluid,conv_fluid_index,heat_capac_rock,conv_rock_index,vel,conv_vel_index);
                    break;
                case 24:
                    perform_convection(i,j,k,i,j+1,k-1,num_rows,num_cols,num_slices,dist_x,dist_y,dist_z,time_inc,temp,next_temp,min_temp_conv,conv_min_temp_index,heat_capac_fluid,conv_fluid_index,heat_capac_rock,conv_rock_index,vel,conv_vel_index);
                    break;
                case 25:
                    perform_convection(i,j,k,i+1,j-1,k-1,num_rows,num_cols,num_slices,dist_x,dist_y,dist_z,time_inc,temp,next_temp,min_temp_conv,conv_min_temp_index,heat_capac_fluid,conv_fluid_index,heat_capac_rock,conv_rock_index,vel,conv_vel_index);
                    break;
                case 26:
                    perform_convection(i,j,k,i+1,j,k-1,num_rows,num_cols,num_slices,dist_x,dist_y,dist_z,time_inc,temp,next_temp,min_temp_conv,conv_min_temp_index,heat_capac_fluid,conv_fluid_index,heat_capac_rock,conv_rock_index,vel,conv_vel_index);
                    break;
                case 27:
                    perform_convection(i,j,k,i+1,j+1,k-1,num_rows,num_cols,num_slices,dist_x,dist_y,dist_z,time_inc,temp,next_temp,min_temp_conv,conv_min_temp_index,heat_capac_fluid,conv_fluid_index,heat_capac_rock,conv_rock_index,vel,conv_vel_index);
                    break;
            }
        }
    }
}

/**
 * Wrapper function for the convection kernel
 */
void convection_cuda() {
    for(int i = 0; i < num_conv_loops; i++) {
        //Calls the convection kernel
        convection_kernel<<<dimGrid,dimBlock>>>(num_cells,num_rows,num_cols,num_slices,dev_dist_x,dev_dist_y,dev_dist_z,time_inc,dev_temp,dev_next_temp,dev_conv_codes,dev_conv_direction,dev_min_temp_conv,dev_conv_min_temp_index,dev_heat_capac_fluid,dev_conv_fluid_index,dev_heat_capac_rock,dev_conv_rock_index,dev_vel,dev_conv_vel_index);
        
        //Waits for the kernel to finish executing
        hipDeviceSynchronize();
        
        //Checks if an error occured during execution
        error = hipGetLastError();
        if(error != hipSuccess) {
			cerr << "Error while executing convection kernel" << endl;
			cerr << error << " : " <<  hipGetErrorString(error) << endl;
			cerr << "Exiting the Program" << endl;
			exit(1);
		}
        
        //Swaps the device temperature arrays
        swap_temp_array_cuda();
    }
}

/**
 * Finds and returns the maximum temperature difference between
 * the current and next temperature arrays.
 */
REAL find_max_temp_diff() {
    REAL max_diff = fabs(next_temp[0] - temp[0]);
    REAL diff = 0.0;
    for(int k = 0; k < num_slices; k++) {
        for(int i = 0; i < num_rows; i++) {
            for(int j = 0; j < num_cols; j++) {
                diff = fabs(next_temp[k*num_rows*num_cols + i*num_cols + j] - temp[k*num_rows*num_cols + i*num_cols + j]);
                if(diff > max_diff) {
                    max_diff = diff;
                }
            }
        }
    }
    return max_diff;
}

/**
 * Finds the index of a given x, y, and z value in meters and
 * stores them in the index array
 */
void find_loc_index(REAL x_loc, REAL y_loc, REAL z_loc, int *index){
	if(x_loc < 0) {
		index[0] = -1;
	}
	else if(x_loc > max_dist_x) {
		index[0] = num_cols;
	}
	else {
		for(index[0] = 0; index[0] < num_cols; index[0]++) {
			if(x_loc <= dist_x[index[0]]+dim_x[index[0]]/2.0) {
				break;
			}
		}
	}

	if(y_loc < 0) {
		index[1] = -1;
	}
	else if(y_loc > max_dist_y) {
		index[1] = num_rows;
	}
	else {
		for(index[1] = 0; index[1] < num_rows; index[1]++) {
			if(y_loc <= dist_y[index[1]]+dim_y[index[1]]/2.0) {
				break;
			}
		}
	}

	if(z_loc < 0) {
		index[2] = -1;
	}
	else if(z_loc > max_dist_z) {
		index[2] = num_slices;
	}
	else {
		for(index[2] = 0; index[2] < num_slices; index[2]++) {
			if(z_loc <= dist_x[index[2]]+dim_x[index[2]]/2.0) {
				break;
			}
		}
	}
	
}


/**
 * Finds the indexes of two corners of the moving source
 * if either falls within the model. The valid parts of the
 * moving source are updated with the moving sources temperature
 */
void update_mvsrc(int index) {
    if(mvsrc_valid[index] == 1) {
        int loc_index[3], loc_offset_index[3];
        find_loc_index(mvsrc_x[index],mvsrc_y[index],mvsrc_z[index],loc_index);
        find_loc_index(mvsrc_x[index]+mvsrc_offset_x[index],mvsrc_y[index]+mvsrc_offset_y[index],mvsrc_z[index]+mvsrc_offset_z[index],loc_offset_index);

        if((loc_index[0] >= 0 && loc_index[0] < num_cols && loc_index[1] >= 0 && loc_index[1] < num_rows && loc_index[2] >= 0 && loc_index[2] < num_slices) || (loc_offset_index[0] >= 0 && loc_offset_index[0] < num_cols && loc_offset_index[1] >= 0 && loc_offset_index[1] < num_rows && loc_offset_index[2] >= 0 && loc_offset_index[2] < num_slices)) {
            for(int k = loc_index[2]; k <=  loc_offset_index[2]; k++) {
                for(int i = loc_index[1]; i <=  loc_offset_index[1]; i++) {
                    for(int j = loc_index[0]; j <=  loc_offset_index[0]; j++) {
                        if(i >= 0 && i < num_rows && j >= 0 && j < num_cols && k >= 0 && k < num_slices) {
                            temp[k*num_rows*num_cols + i*num_cols + j] = mvsrc_temp[index];
                        }
                    }
                }
            }
        }
        else {
            mvsrc_valid[index] = 0;
        }
    }
}

/**
 * Updates the moving sources velocity and position vectors
 * then updates the temperatures in the current temp array
 */
void update_moving_sources() {
    for(int i = 0; i < num_mvsrc; i++) {
        mvsrc_vel_x[i] += mvsrc_accel_x[i]*time_step;
        mvsrc_vel_y[i] += mvsrc_accel_y[i]*time_step;
        mvsrc_vel_z[i] += mvsrc_accel_z[i]*time_step;

        mvsrc_x[i] += mvsrc_vel_x[i]*time_step;
        mvsrc_y[i] += mvsrc_vel_y[i]*time_step;
        mvsrc_z[i] += mvsrc_vel_z[i]*time_step;

        update_mvsrc(i);
    }
}

/**
 * Performs a finite heat flow simulation using
 * conduction and convection.
 */
int main(int argc, char **argv) {

#ifdef DISPLAY
    cout << "\t\t Finite Difference Heat Flow Simulation" << endl;
	//Asks the user if they wish to visualize results
    cout << endl << "Press 1 to run visualization, otherwise 0: ";
    while(!(cin >> display_mode) || display_mode < 0 || display_mode > 1) {
		clear_cin();
        cout << "Incorrect input, to save the state of the model enter 1, else 0: ";
    }
#else
	cout << "\t\t Finite Difference Heat Flow Simulation" << endl;
#endif
    int input_val;    //Temporary int value
    REAL temp_val;    //Temporary REAL value
    
    //Sets the current device
    error = hipSetDevice(0);
    
    //Retrieves the properties of the device
    error = hipGetDeviceProperties(&deviceProp, 0);
    if(error != hipSuccess) {
		cerr << endl << "Error while retrieving device properties" << endl;
		cerr << error << ":" << hipGetErrorString(error) << endl;
		cerr << "Exiting the Program" << endl;
		exit(0);
	}
    
    //Loads the input file for the simulation
    load_file();
    
    //Checks the total and used amount of device global memory after allocation
    size_t free_memory;		//Free memory on the device
    size_t total_memory;	//Total memory on the device
    error = hipMemGetInfo(&free_memory, &total_memory);	//Retrieves the memory information for the device
    if(error != hipSuccess) {
        cerr << endl << "Error while getting memory information" << endl;
		cerr << error << ":" << hipGetErrorString(error) << endl;
		cerr << "Exiting the Program" << endl;
		exit(0);
	}
    cout << "Free memory: "<< (unsigned int)free_memory << ", total memory: "<< (unsigned int)total_memory<<" (after initialization)" << endl;
    
    /**
     * Allows the user to change multiple rectangular blocks of temperatures
     * within the model
     */
     /*
    cout << endl << endl << "To Change the Temp. on a Block, Enter 1, Else 0: ";
    while(!(cin >> input_val) || input_val < 0 || input_val > 1) {
		clear_cin();
        cout << "Incorrect Input, Enter 1 to Change, Else 0: ";
    }
    
    //Warning, the row column pairs need to be space seperated not comma seperated  
    if(input_val == 1) {
        int num_block, row1, row2, col1, col2, slice1, slice2;
        REAL new_temp;
        cout << "Enter the Number of Blocks to Change: ";
        while(!(cin >> num_block) || num_block < 0) {
            clear_cin();
			cout << "Enter a number greater than or equal to 0: ";
		}
        for(int i = 0; i < num_block; i++) {
            cout << endl << "Block " << i << endl;
            cout << "Enter the Coordinates of the Upper Left Corner <row> <column> <slice>: ";
			while(!(cin >> row1 >> col1 >> slice1) || row1 < 0 || col1 < 0 || slice1 < 0) {
                clear_cin();
				cout << "Incorrect input, enter three positive numbers with spaces: ";
			}
            cout << "Enter the Coordinates of the Lower Right Corner <row> <column> <slice>: ";
            while(!(cin >> row2 >> col2 >> slice2) || row2 < row1 || col2 < col1 || slice2 < slice1) {
                clear_cin();
				cout << "Incorrect input, enter three positive numbers with spaces: ";
			}
            cout << endl << "Current Block Temps" << endl;
            cout << setw(10) << "row" << " " << setw(10) << "col" << " " << setw(10) << "slice" << " " << setw(OUT_PRECISION+5) << "temp" << endl;
            cout << setw(10) << row1 << " " << setw(10) << col1 << " " << setw(10) << slice1 << setw(OUT_PRECISION+5) << fixed << setprecision(OUT_PRECISION) << temp[slice1*num_rows*num_cols + row1*num_cols + col1] << endl;
            cout << setw(10) << row2 << " " << setw(10) << col2 << " " << setw(10) << slice2 << setw(OUT_PRECISION+5) << temp[slice2*num_rows*num_cols + row2*num_cols + col2] << endl;
            cout << "Enter a New Temperature For the Block: ";
            while(!(cin >> new_temp)) {
                clear_cin();
                cout << "Incorrect input, enter a new temperature: ";
            }
            for(int i = row1; i < row2; i++) {
                for(int j = col1; j < col2; j++) {
                    for(int k = slice1; k < slice2; k++) {
                        if(i >= 0 && i < num_rows && j >= 0 && j < num_cols && k >= 0 && k < num_slices) {
                            temp[k*num_rows*num_cols + i*num_cols + j] = new_temp;
                        }
                    }
                }
            }
        }
    }
    */
    
    /**
     * Allows the user to start one or more moving sources.
     */
     using_moving_source = 0;
     /*
    cout << endl << endl << "To Start One or More Moving Sources Enter 1, Else Enter 0: ";
    while(!(cin >> using_moving_source) || using_moving_source < 0 || using_moving_source > 1) {
        clear_cin();
        cout << "Incorrect Input, Enter 1 to Change, Else 0: ";
    }
    if(using_moving_source == 1) {
        REAL mag, angle1, angle2;
        cout << "Enter the number of moving sources: ";
        while(!(cin >> num_mvsrc) || num_mvsrc <= 0) {
            clear_cin();
            cout << "Incorrect input, enter a number greater than 0: ";
        }
        mvsrc_x = new REAL[num_mvsrc];
        mvsrc_y = new REAL[num_mvsrc];
        mvsrc_z = new REAL[num_mvsrc];
        mvsrc_offset_x = new REAL[num_mvsrc];
        mvsrc_offset_y = new REAL[num_mvsrc];
        mvsrc_offset_z = new REAL[num_mvsrc];
        mvsrc_vel_x = new REAL[num_mvsrc];
        mvsrc_vel_y = new REAL[num_mvsrc];
        mvsrc_vel_z = new REAL[num_mvsrc];
        mvsrc_accel_x = new REAL[num_mvsrc];
        mvsrc_accel_y = new REAL[num_mvsrc];
        mvsrc_accel_z = new REAL[num_mvsrc];
        mvsrc_temp = new REAL[num_mvsrc];
        mvsrc_valid = new int[num_mvsrc];
        for(int i = 0; i < num_mvsrc; i++) {
            cout << endl << "Moving source " << i << endl;
            cout << "Valid coordinates are x=0-"<<max_dist_x<<" y=0-"<<max_dist_y<<" z=0-"<<max_dist_z<<":" << endl;
            cout << "Enter the coordinates in meters for the corner closest to the origin, <x> <y> <z>: ";
            while(!(cin >> mvsrc_x[i] >> mvsrc_y[i] >> mvsrc_z[i]) || mvsrc_x[i] < 0 || mvsrc_x[i] > max_dist_x || mvsrc_y[i] < 0 || mvsrc_y[i] > max_dist_y || mvsrc_z[i] < 0 || mvsrc_z[i] > max_dist_z) {
                clear_cin();
                cout << "Incorrect input, enter a valid coordinate between x=0-"<<max_dist_x<<" y=0-"<<max_dist_y<<" z=0-"<<max_dist_z<<":";
            }
            cout << "Valid sizes are x=0-"<<max_dist_x-mvsrc_x[i]<<" y=0-"<<max_dist_y-mvsrc_y[i]<<" z=0-"<<max_dist_z-mvsrc_z[i]<<":"<<endl;
            cout << "Enter the size of the moving source in meters, <x size> <y size> <z size>: ";
            while(!(cin >> mvsrc_offset_x[i] >> mvsrc_offset_y[i] >> mvsrc_offset_z[i]) || mvsrc_offset_x[i] <= 0 || mvsrc_offset_x[i] > max_dist_x-mvsrc_x[i] || mvsrc_offset_y[i] <= 0 || mvsrc_offset_y[i] > max_dist_y-mvsrc_y[i] || mvsrc_offset_z[i] <= 0 || mvsrc_offset_z[i] > max_dist_z-mvsrc_z[i]) {
                clear_cin();
                cout << "Incorrect input, enter a valid distance between x=0-"<<max_dist_x-mvsrc_x[i]<<" y=0-"<<max_dist_y-mvsrc_y[i]<<" z=0-"<<max_dist_z-mvsrc_z[i]<<":";
            }
            cout << "Enter the angle of the moving sources vector in degrees from positve x towards negative y (0-360): ";
            while(!(cin >> angle1) || angle1 < 0 || angle1 > 360) {
                clear_cin();
                cout << "Incorrect input, enter a valid angle: ";
            }
            cout << "Enter the angle of the moving sources vector in degrees from positve z (0-180): ";
            while(!(cin >> angle2) || angle2 < 0 || angle2 > 180) {
                clear_cin();
                cout << "Incorrect input, enter a valid angle: ";
            }
            cout << "Enter the magnitude of the velocity vector in m/year: ";
            while(!(cin >> mag) || mag < 0) {
                clear_cin();
                cout << "Incorrect input, enter a velocity greater than 0: ";
            }
            mvsrc_vel_x[i] = mag*sin(angle2/180.0*M_PI)*cos(angle1/180.0*M_PI);
            mvsrc_vel_y[i] = mag*sin(angle2/180.0*M_PI)*sin(angle1/180.0*M_PI);
            mvsrc_vel_z[i] = mag*cos(angle2/180.0*M_PI);

            cout << "Enter the magnitude of the acceleration vector in m/year^2: ";
            while(!(cin >> mag) || mag < 0) {
                clear_cin();
                cout << "Incorrect input, enter an acceleration greater than 0: ";
            }
            mvsrc_accel_x[i] = mag*sin(angle2/180.0*M_PI)*cos(angle1/180.0*M_PI);
            mvsrc_accel_y[i] = mag*sin(angle2/180.0*M_PI)*sin(angle1/180.0*M_PI);
            mvsrc_accel_z[i] = mag*cos(angle2/180.0*M_PI);
            
            cout << "Enter the temperature of the moving source: ";
            while(!(cin >> mag)) {
                clear_cin();
                cout << "Incorrect input, enter a valid temperature: ";
            }
            mvsrc_temp[i] = mag;

            mvsrc_valid[i] = 1;

            update_mvsrc(i);
        }
    }
    */
    //Allows the user to decrease the size of the time step
    cout << endl << endl << "Each Iteration in Time Spans " << scientific << time_step << " Years" << endl;
    cout << "Enter a Shorter Iteration Time in Years if Desired (any larger number otherwise): ";
    while(!(cin >> temp_val) || temp_val <= 0) {
        clear_cin();
        cout << "Incorrect input, enter a number greater than 0: ";
    }
    if(temp_val < time_step) {
        time_step = temp_val;
    }
    
    DHF = chf * QFAC * time_step;

    //Calculates the number of convection loops to perform per time step
    num_conv_loops = (int)(time_step/(10*tic));
    if(num_conv_loops > 5) {
        num_conv_loops = 5;
    }
    else if(num_conv_loops <= 0) {
        num_conv_loops = 1;
    }

    //Calculates the time increment per convection loop
    time_inc = time_step/num_conv_loops;
    min_row_dim = 100.0;
    for(int i = 0; i < num_rows; i++) {
        if(dim_y[i] < min_row_dim) {
            min_row_dim = dim_y[i];
        }
    }
    
    //Asks the user for the runtime of the simulation
    thermal_time_constant = min_row_dim*min_row_dim/max_thermal_conduct_diff;
    cout << endl << endl << "The Thermal Time Constant for the Vertical Dimension is " << thermal_time_constant << " Years" << endl;
    cout << "Enter Time Duration for Calculation in Years: ";
    while(!(cin >> run_time) || run_time <= 0) {
        clear_cin();
        cout << "Incorrect input, enter a number greater than 0: ";
    }
    
    //Asks the user for the number of loops to perform between screen updates
    cout << endl << endl << "Enter the Number of Loops Between Screen Updates: ";
    while(!(cin >> num_loops) || num_loops <= 0) {
        clear_cin();
        cout << "Incorrect input, enter a number greater than 0: ";
    }
    
    use_tolerance = 0;
    /*
    cout << endl << endl << "To have the simulation stop once the temperature change meets a tolerance, Enter 1 otherwise 0: ";
    while(!(cin >> use_tolerance) || use_tolerance < 0 || use_tolerance > 1) {
        clear_cin();
        cout << "Incorrect Input, Enter 1 to use a tolerance, Else 0: ";
    }
    if(use_tolerance == 1) {
        cout << endl << "Enter the tolerance: ";
        while(!(cin >> tolerance) || tolerance  <= 0) {
            clear_cin();
            cout << "Incorrect input, enter a number greater than 0: ";
        }
    }
    */
    //Initializes the simulation time to 0.0
    sim_time = 0.0;
    
    //Waits for the user to hit enter before beginning the simulation
    cout << endl;
    cin.ignore(numeric_limits <streamsize> ::max(), '\n' );
    PressEnterToContinue();
    
    /**
     * The main loop of the simulation
     */
    count = 0;    //Number of loops performed
    cout << endl << endl << num_loops << " loops between screen updates" << endl << endl;
    if(use_tolerance == 0) {
        cout << setw(15) << "num loops" << setw(20) << "run time (years)" << setw(20) << "sim time (years)" << endl;
    }
    else {
        cout << setw(15) << "num loops" << setw(20) << "run time (years)" << setw(20) << "sim time (years)" << setw(20) << "Max temp diff" << endl;
    }

#ifdef DISPLAY
	if(display_mode == 1) {
		array_minmax();
		array_size = num_cols * num_rows;
		color_field = new float[array_size * 3];
		for (int i=0; i<array_size *3; i++) {
			color_field[i] = 0.0;
		}

		glutInit(&argc, argv);
		int windowWidth = glutGet(GLUT_SCREEN_WIDTH);
		int windowHeight = glutGet(GLUT_SCREEN_HEIGHT);

		glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE | GLUT_DEPTH);
		glutInitWindowSize(windowWidth, windowHeight);
		glutInitWindowPosition(0, 0);
		
		glutCreateWindow("ARC Simulation");

		glViewport(0, 0, windowWidth,windowHeight);

		glEnable (GL_BLEND);
		glBlendFunc (GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);

		glMatrixMode(GL_PROJECTION);
		glLoadIdentity();
		gluPerspective(60, 1.77777f, 1.0, 20000.0);

		glutDisplayFunc(display3D);
		
		//glutMouseFunc(mouse_button);//Mouse motion and camera trans settings maintained for debugging
		//glutMotionFunc(mouse_move);
		glutKeyboardFunc(keyboard);
		glutSpecialFunc(keyboardSpecial);

		/*camera_trans[0] = -num_cols/2.0;
		camera_trans[1] = num_rows/3.0;
		camera_trans[2] = -num_rows*1.75*tan(28.0/180.0*M_PI);
		camera_rot[0] = 28.0;
		camera_trans_lag[0] = -num_cols/2.0;
		camera_trans_lag[1] = num_rows/3.0;
		camera_trans_lag[2] = -num_rows*1.75*tan(28.0/180.0*M_PI);
		camera_rot_lag[0] = 28.0;
		*/

		//gluLookAt(num_cols/2.0,num_rows*0.1,num_rows,num_cols/2.0,-num_rows/3.0,0.0,0.0,1.0,0.0);
		if(num_rows > num_cols) {
            gluLookAt(num_cols/2.0,num_rows*0.1,num_rows,num_cols/2.0,-num_rows/3.0,0.0,0.0,1.0,0.0);
        }
        else {
            gluLookAt(num_cols/2.0,num_rows*0.1,num_cols,num_cols/2.0,-num_rows/3.0,0.0,0.0,1.0,0.0);
        }
        glutMainLoop();
		PressEnterToContinue();
	}
	else {
#endif
        while(sim_time <= run_time) {
            //Displays status information for the current loop
            if(count%num_loops == 0) {
                if(use_tolerance == 0) {
                    cout << setw(15) << count << setw(20) << fixed << setprecision(5) << sim_time << setw(20) << initial_time + sim_time << endl;
                }
                else {
                    cout << setw(15) << count << setw(20) << fixed << setprecision(5) << sim_time << setw(20) << initial_time + sim_time << setw(20) << max_temp_diff << endl;
                }
                //Saves the current state of the simulation if the save_state flag is set
                if(save_state) {
                    save_model_state();
                }
            }
            
            //Performs convection updates if the current simulation is using convection
            if(using_convection) {
                convection_cuda();
            }
            
            //Performs conduction calculations
            conduction_cuda();
            
            //Increments the simulation time and loop count
            sim_time += time_step;
            count++;
            
            if(use_tolerance == 1) {
                max_temp_diff = find_max_temp_diff();
                if(max_temp_diff < tolerance) {
                    cout << "Maximum temperature change below the tolerance, stoping the simulation" << endl;
                    break;
                }
            }

            //Updates the moving source
            if(using_moving_source == 1) {
                update_moving_sources();
            }
        }
        
        //Saves the final result of the simulation
        if(save_state == 1 || save_result == 1) {
            save_model_state();
        }
        save_surfer();

        //Waits for the user to hit enter before ending the simulation
        cout << endl << "Simulation Complete" << endl;
        PressEnterToContinue();

        deallocate_memory();
        deallocate_cuda_memory();
#ifdef DISPLAY
	}
#endif
}
